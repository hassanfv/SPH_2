#include "hip/hip_runtime.h"
//%%writefile test.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <random>
#include "myCppSPHLibs_v7_t6_exclude_BH.h"
#include <hiprand/hiprand_kernel.h>
using namespace std;

// smoothing func. updated + my_pi is replaced by M_PI. (28 Jan - 2023)
// In this version, we implemented the Restart possibility! (24 Jan - 2023)
// In this version, we use CLOUDY cooling & heating!
// In this version, we also include cooling.
// In this version, the output file also contains the velocity components.

string RESTART = "no"; // options = yes or no (note they are in lower case letters)!!!!!!!!!!!!!!!!!



// Function to save the OUTPUT Snap-Shots!!
void saveArraysToBinary(const std::string& filename, float* x, float* y, float* z, float* vx, float* vy, float* vz, int N, float* rho, float* h, float* u, int NG)
{
    // Open the file in binary mode
    std::ofstream file(filename, std::ios::binary);
    
    // Check if the file was opened successfully
    if (!file)
    {
        std::cerr << "Failed to open file for writing: " << filename << std::endl;
        return;
    }
    
    // Write N and NG to the file
    file.write(reinterpret_cast<const char*>(&N), sizeof(int));
    file.write(reinterpret_cast<const char*>(&N), sizeof(int));
    
    // Write the arrays to the file
    file.write(reinterpret_cast<const char*>(x), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(y), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(z), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(vx), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(vy), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(vz), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(rho), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(h), N * sizeof(float));
    file.write(reinterpret_cast<const char*>(u), N * sizeof(float));
    
    // Close the file
    file.close();
}



//========================================
//======= Smoothing Length (BH) =========
//========================================

float smoothing_h_BH(float *x, float *y, float *z, float hBH,
                                int N, int Ndown, int Nup, float coeff)
{

  float h_new = 2.0f * hBH;
  float h_tmp = h_new;
  int N_iter = 0;
  int k = 0;
  
  float xBH = 0.0f;
  float yBH = 0.0f;
  float zBH = 0.0f;

  float dx, dy, dz;
  while ((k < Ndown) || (k > Nup))
  {

    k = 0;

    for (int j = 0; j < N; j++)
    {
      dx = x[j] - xBH;
      dy = y[j] - yBH;
      dz = z[j] - zBH;
      float rr = sqrt(dx * dx + dy * dy + dz * dz);

      if (rr <= h_new)
      {
        k++;
      }
    }

    if (k < Ndown)
    {
      h_new = h_new + coeff * 2.0f * hBH;
    }

    if (k > Nup)
    {
      h_new = h_new - coeff * 2.0f * hBH;
    }

    if (h_new > h_tmp)
    {
      h_tmp = h_new;
    }

    N_iter++;
    if (N_iter > 1000)
    {
      h_new = h_tmp;
      break;
    }
  }
  hBH = 0.5f * h_new;
  return hBH;
}







//---------------------
float max_finder(float *arr, int N)
{

  float max_val = 0.0;
  for (int i = 0; i < N; i++)
  {
    if (arr[i] >= max_val)
    {
      max_val = arr[i];
    }
  }
  return max_val;
}

float min_finder(float *arr, int N)
{

  float min_val = arr[0];
  for (int i = 0; i < N; i++)
  {
    if (arr[i] <= min_val)
    {
      min_val = arr[i];
    }
  }
  return min_val;
}

int main()
{

  float dt = 1e-4; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! This is only the first time step !!

  const int Nngb_f = 64.0f; // used in smoothing func.
  const int Nngb = 64;
  const int Ndown = Nngb - 5;
  const int Nup = Nngb + 5;
  const float coeff = 0.001f; // used for smoothing length.

  //***************************************
  //********** Reading params.txt file ************
  //***************************************

  // Open the file
  std::ifstream filex("params.txt");

  // Check if file was successfully opened
  if (!filex) {
    std::cerr << "Unable to open file param.txt";
    return 1; // return with error code 1
  }

  // Variables to store the values
  int N_tot, N, N_blank;
  float G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res;

  // Read the values from the file
  filex >> N_tot;
  filex >> N;
  filex >> N_blank;
  filex >> G;
  filex >> L_AGN_code_unit;
  filex >> M_dot_in;
  filex >> v_in;
  filex >> u_for_10K_Temp;
  filex >> m_sph_high_res;

  // Close the file
  filex.close();


  int N_real = N;
  
  N = N_tot;


  //***************************************
  //********** Reading IC file ************
  //***************************************

  std::vector<float> xvec(N_tot);
  std::vector<float> yvec(N_tot);
  std::vector<float> zvec(N_tot);
  std::vector<float> vxvec(N_tot);
  std::vector<float> vyvec(N_tot);
  std::vector<float> vzvec(N_tot);
  std::vector<float> massvec(N_tot);
  std::vector<float> hpreviousvec(N_tot);
  std::vector<float> epsvec(N_tot);
  std::vector<float> uvec(N_tot);

  // Check if the binary file exists
  std::ifstream file("IC_Richings2018_110k.bin", std::ios::binary);
  if (!file) {
    std::cout << "Could not open the binary file." << std::endl;
  }
  file.close();
  
  // Open the binary file for reading
  file.open("IC_Richings2018_110k.bin", std::ios::binary);
  if (file) {
    // Read the first array
    file.read(reinterpret_cast<char*>(xvec.data()), sizeof(float) * xvec.size());
    file.read(reinterpret_cast<char*>(yvec.data()), sizeof(float) * yvec.size());
    file.read(reinterpret_cast<char*>(zvec.data()), sizeof(float) * zvec.size());
    
    file.read(reinterpret_cast<char*>(vxvec.data()), sizeof(float) * vxvec.size());
    file.read(reinterpret_cast<char*>(vyvec.data()), sizeof(float) * vyvec.size());
    file.read(reinterpret_cast<char*>(vzvec.data()), sizeof(float) * vzvec.size());
    
    file.read(reinterpret_cast<char*>(massvec.data()), sizeof(float) * massvec.size());
    file.read(reinterpret_cast<char*>(hpreviousvec.data()), sizeof(float) * hpreviousvec.size());
    file.read(reinterpret_cast<char*>(epsvec.data()), sizeof(float) * epsvec.size());
    file.read(reinterpret_cast<char*>(uvec.data()), sizeof(float) * uvec.size());

    // Close the file
    file.close();

  } else {
    std::cout << "Failed to open the file." << std::endl;
  }




  float ttime;

  // declaring the arrays.
  float *x, *d_x, *y, *d_y, *z, *d_z, *vx, *d_vx, *vy, *d_vy, *vz, *d_vz;
  float *mass, *d_mass, *h, *d_h, *hprevious, *d_hprevious, *rho, *d_rho;
  float *accx, *accy, *accz, *d_accx, *d_accy, *d_accz, *eps, *d_eps;
  float *P, *d_P, *csnd, *d_csnd, *divV, *d_divV, *curlV, *d_curlV;
  float *accx_sph, *accy_sph, *accz_sph, *d_accx_sph, *d_accy_sph, *d_accz_sph;
  float *accx_tot, *accy_tot, *accz_tot, *d_accx_tot, *d_accy_tot, *d_accz_tot;
  float *abs_acc_g, *abs_acc_tot, *v_sig, *dh_dt, *d_abs_acc_g, *d_abs_acc_tot;
  float *d_v_sig, *d_dh_dt, *u, *dudt, *d_u, *d_dudt, *uprevious, *utprevious;
  float *d_uprevious, *d_utprevious;
  float *Nngb_previous, *d_Nngb_previous; // Note that both are floats and not int! check smoothing func. to see why!
  float *weightsBH, *d_weightsBH, *dt_particles, *d_dt_particles;

  float gammah = 5.0f/3.0f;

  x = new float[N];
  y = new float[N];
  z = new float[N];

  vx = new float[N];
  vy = new float[N];
  vz = new float[N];

  accx = new float[N];
  accy = new float[N];
  accz = new float[N];

  mass = new float[N];
  h = new float[N];
  hprevious = new float[N];
  rho = new float[N];
  eps = new float[N];
  P = new float[N];
  csnd = new float[N];

  divV = new float[N];
  curlV = new float[N];

  accx_sph = new float[N];
  accy_sph = new float[N];
  accz_sph = new float[N];

  accx_tot = new float[N];
  accy_tot = new float[N];
  accz_tot = new float[N];

  abs_acc_g = new float[N];
  abs_acc_tot = new float[N];
  v_sig = new float[N];
  dh_dt = new float[N];
  dt_particles = new float[N];

  u = new float[N];
  dudt = new float[N];
  uprevious = new float[N];
  utprevious = new float[N];

  Nngb_previous = new float[N];

  weightsBH = new float[N];

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  hipMalloc(&d_vx, N * sizeof(float));
  hipMalloc(&d_vy, N * sizeof(float));
  hipMalloc(&d_vz, N * sizeof(float));

  hipMalloc(&d_accx, N * sizeof(float));
  hipMalloc(&d_accy, N * sizeof(float));
  hipMalloc(&d_accz, N * sizeof(float));

  hipMalloc(&d_mass, N * sizeof(float));
  hipMalloc(&d_h, N * sizeof(float));
  hipMalloc(&d_hprevious, N * sizeof(float));
  hipMalloc(&d_rho, N * sizeof(float));
  hipMalloc(&d_eps, N * sizeof(float));
  hipMalloc(&d_P, N * sizeof(float));
  hipMalloc(&d_csnd, N * sizeof(float));

  hipMalloc(&d_divV, N * sizeof(float));
  hipMalloc(&d_curlV, N * sizeof(float));

  hipMalloc(&d_accx_sph, N * sizeof(float));
  hipMalloc(&d_accy_sph, N * sizeof(float));
  hipMalloc(&d_accz_sph, N * sizeof(float));

  hipMalloc(&d_accx_tot, N * sizeof(float));
  hipMalloc(&d_accy_tot, N * sizeof(float));
  hipMalloc(&d_accz_tot, N * sizeof(float));

  hipMalloc(&d_abs_acc_g, N * sizeof(float));
  hipMalloc(&d_abs_acc_tot, N * sizeof(float));
  hipMalloc(&d_v_sig, N * sizeof(float));
  hipMalloc(&d_dh_dt, N * sizeof(float));
  hipMalloc(&d_dt_particles, N * sizeof(float));

  hipMalloc(&d_u, N * sizeof(float));
  hipMalloc(&d_dudt, N * sizeof(float));
  hipMalloc(&d_uprevious, N * sizeof(float));
  hipMalloc(&d_utprevious, N * sizeof(float));

  hipMalloc(&d_Nngb_previous, N * sizeof(float));

  hipMalloc(&d_weightsBH, N * sizeof(float));

  /* IC file
  0  1  2  3   4   5   6  7          8    9
  x, y, z, vx, vy, vz, m, hprevious, eps, u
  */

  /* RESTART file
  0  1  2  3   4   5   6  7          7    9   10
  x, y, z, vx, vy, vz, m, hprevious, eps, u,  dudt (u wil be uprevious and dudt will be utprevious)!
  */

  // Initialize x, y, and z on the Host (All particles, i.e. Gas + DM + BH).
  for (int i = 0; i < N; i++)
  {
    x[i] = xvec[i];
    y[i] = yvec[i];
    z[i] = zvec[i];

    vx[i] = vxvec[i];
    vy[i] = vyvec[i];
    vz[i] = vzvec[i];

    mass[i] = massvec[i];
    eps[i] = epsvec[i];

    accx[i] = 0.0f;
    accy[i] = 0.0f;
    accz[i] = 0.0f;

    accx_tot[i] = 0.0f;
    accy_tot[i] = 0.0f;
    accz_tot[i] = 0.0f;

    abs_acc_g[i] = 0.0f;
    abs_acc_tot[i] = 0.0f;
    v_sig[i] = 0.0f;
    //NGroupz[i] = i;
    
  }
  
  
  // Initialize x, y, and z on the Host (Only Gas particles).
  for (int i = 0; i < N; i++)
  {
    hprevious[i] = hpreviousvec[i];
    h[i] = hpreviousvec[i];    // place holder.
    rho[i] = 0.0f;  // place holder.
    P[i] = 0.0f;    // placeholder.
    csnd[i] = 0.0f; // placeholder.

    divV[i] = 0.0f;  // placeholder.
    curlV[i] = 0.0f; // placeholder.

    accx_sph[i] = 0.0f;
    accy_sph[i] = 0.0f;
    accz_sph[i] = 0.0f;

    dh_dt[i] = 0.0f;

    u[i] = uvec[i];
    dudt[i] = 0.0f;
    uprevious[i] = 0.0f;
    utprevious[i] = 0.0f;
    
    dt_particles[i] = 0.0f;

    Nngb_previous[i] = Nngb_f;
    
    weightsBH[i] = 0.0f;
    
  }

  // Copy from Host to Device.
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_vx, vx, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vy, vy, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vz, vz, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx, accx, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy, accy, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz, accz, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_mass, mass, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_h, h, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_hprevious, hprevious, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rho, rho, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_eps, eps, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_P, P, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_csnd, csnd, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_divV, divV, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_curlV, curlV, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx_sph, accx_sph, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy_sph, accy_sph, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz_sph, accz_sph, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx_tot, accx_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy_tot, accy_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz_tot, accz_tot, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_abs_acc_g, abs_acc_g, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_abs_acc_tot, abs_acc_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_sig, v_sig, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dh_dt, dh_dt, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dt_particles, dt_particles, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_u, u, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dudt, dudt, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_uprevious, uprevious, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_utprevious, utprevious, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_Nngb_previous, Nngb_previous, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_weightsBH, weightsBH, N * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;                            // number of threads in a block
  int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks in a grid

  const float visc_alpha = 1.0f;
  //const float mH = 1.6726e-24; // gram
  //const float kB = 1.3807e-16; // cm2 g s-2 K-1
  //const float XH = 0.76;

  // We set MAX_dt_code_unit to avoid negative u !
  // float MAX_dt_code_unit = ref_dt_cgs / unitTime_in_s;



  float t;

  if (RESTART == "no")
  {
    t = 0.0f;
  }
  else
  {
    t = ttime;
  }

  // float dt = MAX_dt_code_unit;
  float tEnd = 5.0f;
  float Nt = ceil(tEnd / dt) + 1;
  
  N = N_real; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //-----------------------------------------------
  //-------------- Smoothing Length ---------------
  //-----------------------------------------------
  if (RESTART == "no")
  {
    smoothing_h_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_h, d_hprevious,
                                         N, Ndown, Nup, coeff,
                                         Nngb_f, d_Nngb_previous, d_divV, dt);
    hipDeviceSynchronize();
  }

  //-----------------------------------------------
  //----------------- getDensity ------------------
  //-----------------------------------------------
  getDensity_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_mass,
                                      d_rho, d_h, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------ getAcc_g -------------------
  //-----------------------------------------------
  acc_g<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_eps, d_accx, d_accy, d_accz,
                                 d_mass, G, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //---------------- getPressure ------------------
  //-----------------------------------------------
  getPressure_Adiabatic_exBH<<<gridSize, blockSize>>>(d_P, d_rho, d_u, gammah, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- getCsound -------------------
  //-----------------------------------------------
  getCsound_Adiabatic_exBH<<<gridSize, blockSize>>>(d_csnd, d_u, gammah, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- div_curlV -------------------
  //-----------------------------------------------
  div_curlVel_exBH<<<gridSize, blockSize>>>(d_divV, d_curlV, d_x, d_y, d_z, d_vx, d_vy, d_vz,
                                       d_rho, d_mass, d_h, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------ acc_sph --------------------
  //-----------------------------------------------
  acc_sph_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                   d_divV, d_curlV, d_mass, d_P, d_accx_sph, d_accy_sph,
                                   d_accz_sph, visc_alpha, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------ acc_tot --------------------
  //-----------------------------------------------
  acc_g_sph<<<gridSize, blockSize>>>(d_accx_tot, d_accy_tot, d_accz_tot,
                                            d_accx, d_accy, d_accz,
                                            d_accx_sph, d_accy_sph, d_accz_sph,
                                            N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------- du_dt ---------------------
  //-----------------------------------------------
  if (RESTART == "no")
  {
    get_dU_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                    d_divV, d_curlV, d_mass, d_P, d_dudt,
                                    visc_alpha, N);
    hipDeviceSynchronize();
  }

  //-----------------------------------------------
  //---------------- u evolution ------------------
  //-----------------------------------------------
  if (RESTART == "no")
  {
    u_updater1_exBH<<<gridSize, blockSize>>>(d_u, d_dudt, dt, N);
    hipDeviceSynchronize();
  }


  //-----------------------------------------------
  //-------- updating uprevious, utprevious -------
  //-----------------------------------------------
  if (RESTART == "no")
  {
    u_ut_previous_updater_exBH<<<gridSize, blockSize>>>(d_u, d_dudt, d_uprevious,
                                                   d_utprevious, N);
    hipDeviceSynchronize();
  }

  const float C_CFL = 0.25;

  float dt_max;
  //float dt_B0, dt_B1, dt_B2, dt_B3;
  //int jG1, jG2, jG3, NG1, NG2, NG3, NN, jj;
  //NG1 = NG; // Note that NG1, NG2, and NG3 are just accidentally named like NG!!!
  //jj = 1;

  float h_min, h_max;
  
  float leftover_mass = 0.0f;

  //int Energy_checker = 0;


  // **************************************************************
  // *********************** MAIN LOOP ****************************
  // **************************************************************

  int counter = 0;

  while (t < tEnd)
  {

    auto begin = std::chrono::high_resolution_clock::now();

    //****************** velocity evolution *******************
    v_evolve<<<gridSize, blockSize>>>(d_vx, d_vy, d_vz, d_accx_tot, d_accy_tot,
                                      d_accz_tot, dt, N);
    hipDeviceSynchronize();  

    //****************** position evolution (BH fixed at [0, 0, 0]) *******************

    r_evolve_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, dt, N);
    hipDeviceSynchronize();

    //****************** Smoothing Length *********************

    smoothing_h_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_h, d_hprevious,
                                          N, Ndown, Nup, coeff,
                                          Nngb_f, d_Nngb_previous, d_divV,
                                          dt);
    hipDeviceSynchronize();


    //****************** updating hprevious ***************
    hprevious_updater<<<gridSize, blockSize>>>(d_hprevious,
                                               d_h, N);
    hipDeviceSynchronize();

    //****************** getDensity ***********************
    getDensity_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_mass,
                                        d_rho, d_h, N);
    hipDeviceSynchronize();

    //****************** getAcc_gX *************************
    auto T_acc_g = std::chrono::high_resolution_clock::now();
    acc_g<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_eps, d_accx, d_accy, d_accz,
                                    d_mass, G, N);
    hipDeviceSynchronize();
    auto end_acc_g = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_g = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_g - T_acc_g);
    cout << "T_acc_g = " << elapsed_acc_g.count() * 1e-9 << endl;

    //****************** getPressure **********************
    getPressure_Adiabatic_exBH<<<gridSize, blockSize>>>(d_P, d_rho, d_u, gammah, N);
    hipDeviceSynchronize();

    //****************** getCsound ************************
    getCsound_Adiabatic_exBH<<<gridSize, blockSize>>>(d_csnd, d_u, gammah, N);
    hipDeviceSynchronize();

    //****************** div_curlVX ************************ 
    auto T_divCurl = std::chrono::high_resolution_clock::now();
    div_curlVel_exBH<<<gridSize, blockSize>>>(d_divV, d_curlV, d_x, d_y, d_z, d_vx, d_vy, d_vz,
                                          d_rho, d_mass, d_h, N);
    hipDeviceSynchronize();
    auto end_divCurl = std::chrono::high_resolution_clock::now();
    auto elapsed_divCurl = std::chrono::duration_cast<std::chrono::nanoseconds>(end_divCurl - T_divCurl);
    cout << "T_divCurl = " << elapsed_divCurl.count() * 1e-9 << endl;

    //****************** acc_sphX ************************** 
    auto T_acc_sph = std::chrono::high_resolution_clock::now();
    acc_sph_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                      d_divV, d_curlV, d_mass, d_P, d_accx_sph, d_accy_sph,
                                      d_accz_sph, visc_alpha, N);
    hipDeviceSynchronize();
    auto end_acc_sph = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_sph = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_sph - T_acc_sph);
    cout << "T_acc_sph = " << elapsed_acc_sph.count() * 1e-9 << endl;

    //****************** acc_tot **************************
    auto T_acc_tot = std::chrono::high_resolution_clock::now();
    acc_g_sph_exBH<<<gridSize, blockSize>>>(d_accx_tot, d_accy_tot, d_accz_tot,
                                       d_accx, d_accy, d_accz,
                                       d_accx_sph, d_accy_sph, d_accz_sph, N);
    hipDeviceSynchronize();
    auto end_acc_tot = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_tot = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_tot - T_acc_tot);
    cout << "T_acc_tot = " << elapsed_acc_tot.count() * 1e-9 << endl;

    //****************** velocity evolution *******************
    v_evolve<<<gridSize, blockSize>>>(d_vx, d_vy, d_vz, d_accx_tot, d_accy_tot,
                                      d_accz_tot, dt, N);
    hipDeviceSynchronize();

    //******************** get_dUX (du_dt) *********************
    auto T_dU = std::chrono::high_resolution_clock::now();
    get_dU_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                     d_divV, d_curlV, d_mass, d_P, d_dudt,
                                     visc_alpha, N);
    hipDeviceSynchronize();
    auto end_dU = std::chrono::high_resolution_clock::now();
    auto elapsed_dU = std::chrono::duration_cast<std::chrono::nanoseconds>(end_dU - T_dU);
    cout << "T_dU = " << elapsed_dU.count() * 1e-9 << endl;

    //******************** u evolution *********************
    u_updater_main_exBH<<<gridSize, blockSize>>>(d_u, d_dudt, d_uprevious,
                                            d_utprevious, dt, N);
    hipDeviceSynchronize();




    
    
    //**************** AGN Wind Injection *******************
    agn_wind_injection<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, 
                                                d_u, d_mass, d_h, d_eps, M_dot_in, v_in, dt,
                                                leftover_mass, m_sph_high_res, u_for_10K_Temp, N);
    
    N = N + static_cast<int>(u[0]); // Updating the number of particles as new particles may be created above!
    leftover_mass = u[N+1];


    hipMemcpy(u, d_u, N*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++)
    {
      cout << "uuuuu = " << u[i] << endl;
    }


    //-------------------------------------------------

    h_min = min_finder(h, N);
    h_max = max_finder(h, N);
    cout << "min_h = " << h_min << endl;
    cout << "max_h = " << h_max << endl;

    

    hipMemcpy(rho, d_rho, N*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++)
    {
      cout << "AAA = " << rho[i] << endl;
    }



    
    //------------ SAVING SNAP-SHOTS ------------
    if(!(counter % 20)){
      hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(vx, d_vx, N*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vy, d_vy, N*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vz, d_vz, N*sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(rho, d_rho, N*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(h, d_h, N*sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(u, d_u, N*sizeof(float), hipMemcpyDeviceToHost);

      // Specify the output file name
      std::string filename = "./Outputs/G-"+ to_string(t*1) + ".bin";
      // Save the arrays to binary format
      saveArraysToBinary(filename, x, y, z, vx, vy, vz, N, rho, h, u, N);
    }

    //******* updating uprevious, utprevious ********
    auto T_u_ut = std::chrono::high_resolution_clock::now();
    u_ut_previous_updater_exBH<<<gridSize, blockSize>>>(d_u, d_dudt, d_uprevious,
                                                   d_utprevious, N);
    hipDeviceSynchronize();
    auto end_u_ut = std::chrono::high_resolution_clock::now();
    auto elapsed_u_ut = std::chrono::duration_cast<std::chrono::nanoseconds>(end_u_ut - T_u_ut);
    cout << "T_update_u_ut = " << elapsed_u_ut.count() * 1e-9 << endl;


    //auto end_FOR = std::chrono::high_resolution_clock::now();
    //auto elapsed_FOR = std::chrono::duration_cast<std::chrono::nanoseconds>(end_FOR - T_FOR);
    //cout << "T_FOR = " << elapsed_FOR.count() * 1e-9 << endl;

    auto T_NG = std::chrono::high_resolution_clock::now();
    //hipMemcpy(d_NGroupz, NGroupz, NG * sizeof(int), hipMemcpyHostToDevice);
    auto end_NG = std::chrono::high_resolution_clock::now();
    auto elapsed_NG = std::chrono::duration_cast<std::chrono::nanoseconds>(end_NG - T_NG);
    cout << "T_NG = " << elapsed_NG.count() * 1e-9 << endl;

 
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

    cout << "Elapsed time = " << elapsed.count() * 1e-9 << endl;
    cout << endl;

    
    //******************************************************
    //************* Updating Time-step dt ******************
    //******************************************************
    
    dt_array_indiv_dt_exBH<<<gridSize, blockSize>>>(d_x, d_y, d_z,
                                               d_vx, d_vy, d_vz,
                                               d_accx, d_accy, d_accz,
                                               d_accx_tot, d_accy_tot, d_accz_tot,
                                               d_h, d_csnd, d_dt_particles,
                                               d_abs_acc_g, d_abs_acc_tot,
                                               d_divV, d_dh_dt, C_CFL,
                                               visc_alpha, d_eps, N);
    hipDeviceSynchronize();

    hipMemcpy(dt_particles, d_dt_particles, N * sizeof(float), hipMemcpyDeviceToHost);

    dt = min_finder(dt_particles, N);

    dt_max = max_finder(dt_particles, N);
    
    
    

    t += dt;

    if (!(counter % 1))
    {
      cout << "Adopted dt = " << dt << endl;
      cout << "current t = " << t << endl;
      cout << "*****************************" << endl;
      cout << endl;
    }

    counter++;
  }

  delete[] x;
  delete[] y;
  delete[] z;
  delete[] vx;
  delete[] vy;
  delete[] vz;
  delete[] mass;
  delete[] h;
  delete[] hprevious;
  delete[] rho;
  delete[] accx;
  delete[] accy;
  delete[] accz;
  delete[] eps;
  delete[] P;
  delete[] csnd;
  delete[] divV;
  delete[] curlV;
  delete[] accx_sph;
  delete[] accy_sph;
  delete[] accz_sph;
  delete[] accx_tot;
  delete[] accy_tot;
  delete[] accz_tot;
  delete[] abs_acc_g;
  delete[] abs_acc_tot;
  delete[] v_sig;
  delete[] dh_dt;
  delete[] u;
  delete[] dudt;
  delete[] uprevious;
  delete[] utprevious;

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_vx);
  hipFree(d_vy);
  hipFree(d_vz);
  hipFree(d_mass);
  hipFree(d_h);
  hipFree(d_hprevious);
  hipFree(d_rho);
  hipFree(d_accx);
  hipFree(d_accy);
  hipFree(d_accz);
  hipFree(d_P);
  hipFree(d_csnd);
  hipFree(d_divV);
  hipFree(d_curlV);
  hipFree(d_accx_sph);
  hipFree(d_accy_sph);
  hipFree(d_accz_sph);
  hipFree(d_accx_tot);
  hipFree(d_accy_tot);
  hipFree(d_accz_tot);
  hipFree(d_abs_acc_g);
  hipFree(d_abs_acc_tot);
  hipFree(d_v_sig);
  hipFree(d_dh_dt);
  hipFree(d_u);
  hipFree(d_dudt);
  hipFree(d_uprevious);
  hipFree(d_utprevious);
}
