#include "hip/hip_runtime.h"
//%%writefile test.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <random>
#include <tuple>

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include "multi_libs.h"

using namespace std;

// Function to save results
void saveArraysToBinary(const std::string &filename, float *h, size_t N)
{
  std::ofstream out(filename, std::ios::binary);
  out.write(reinterpret_cast<const char *>(h), N * sizeof(float));
  out.close();
}

int main()
{

  float dt = 1e-7; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! This is only the first time step !!

  const int Nngb_f = 64.0f; // used in smoothing func.
  const int Nngb = 64;
  const int Ndown = Nngb - 5;
  const int Nup = Nngb + 5;
  const float coeff = 0.001f; // used for smoothing length.
  float gammah = 5.0f / 3.0f;
  const float C_CFL = 0.25;

  float h_min, h_max, h_mean;

  // Reading the params.txt file
  std::string filename;
  int N;
  float G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma;

  readParams(filename, N, G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma);

  std::cout << "filename: " << filename << "\n";
  std::cout << "N: " << N << "\n";
  std::cout << "G: " << G << "\n";
  std::cout << "L_AGN_code_unit: " << L_AGN_code_unit << "\n";
  std::cout << "M_dot_in_code_unit: " << M_dot_in << "\n";
  std::cout << "vin_in_code_unit: " << v_in << "\n";
  std::cout << "u_for_10K_Temp: " << u_for_10K_Temp << "\n";
  std::cout << "m_sph_high_res: " << m_sph_high_res << "\n";
  std::cout << "sigma: " << sigma << "\n";

  // Reading the IC file
  auto data = readVectorsFromFile(N, filename);

  std::vector<int> &Typvec = std::get<0>(data);
  std::vector<float> &xvec = std::get<1>(data);
  std::vector<float> &yvec = std::get<2>(data);
  std::vector<float> &zvec = std::get<3>(data);
  std::vector<float> &vxvec = std::get<4>(data);
  std::vector<float> &vyvec = std::get<5>(data);
  std::vector<float> &vzvec = std::get<6>(data);
  std::vector<float> &massvec = std::get<7>(data);
  std::vector<float> &hvec = std::get<8>(data);
  std::vector<float> &epsvec = std::get<9>(data);
  std::vector<float> &uvec = std::get<10>(data);

  // Allocate and initialize host memory for full arrays
  int *Typ = new int[N];

  float *x = new float[N];
  float *y = new float[N];
  float *z = new float[N];

  float *vx = new float[N];
  float *vy = new float[N];
  float *vz = new float[N];

  float *h = new float[N];
  float *eps = new float[N];
  float *mass = new float[N];
  float *u = new float[N];

  float *accx = new float[N];
  float *accy = new float[N];
  float *accz = new float[N];

  float *accx_tot = new float[N];
  float *accy_tot = new float[N];
  float *accz_tot = new float[N];

  float *abs_acc_g = new float[N];
  float *abs_acc_tot = new float[N];
  float *v_sig = new float[N];

  float *rho = new float[N];
  float *P = new float[N];
  float *csnd = new float[N];

  float *divV = new float[N];
  float *curlV = new float[N];

  float *accx_sph = new float[N];
  float *accy_sph = new float[N];
  float *accz_sph = new float[N];

  float *dh_dt = new float[N];

  float *dudt = new float[N];
  float *utprevious = new float[N];

  float *dt_particles = new float[N];

  float *Nngb_previous = new float[N];

  for (int i = 0; i < N; i++)
  {

    if (i == 200000)
      cout << "check" << endl;

    Typ[i] = Typvec[i];

    x[i] = xvec[i];
    y[i] = yvec[i];
    z[i] = zvec[i];

    vx[i] = vxvec[i];
    vy[i] = vyvec[i];
    vz[i] = vzvec[i];

    h[i] = hvec[i];
    eps[i] = epsvec[i];
    mass[i] = massvec[i];
    u[i] = uvec[i];

    accx[i] = 0.0f;
    accy[i] = 0.0f;
    accz[i] = 0.0f;

    accx_tot[i] = 0.0f;
    accy_tot[i] = 0.0f;
    accz_tot[i] = 0.0f;

    abs_acc_g[i] = 0.0f;
    abs_acc_tot[i] = 0.0f;
    v_sig[i] = 0.0f;

    rho[i] = 0.0f;  // place holder.
    P[i] = 0.0f;    // placeholder.
    csnd[i] = 0.0f; // placeholder.

    divV[i] = 0.0f;  // placeholder.
    curlV[i] = 0.0f; // placeholder.

    if (Typ[i] == 0)
    {
      Nngb_previous[i] = Nngb_f;
    }
    else
    {
      Nngb_previous[i] = 0.0f;
    }
  }

  int nGPUs;
  hipGetDeviceCount(&nGPUs);

  cout << "nGPUs = " << nGPUs << endl;

  // Pointers for device memory
  int *d_Typ[nGPUs];
  float *d_x[nGPUs], *d_y[nGPUs], *d_z[nGPUs], *d_eps[nGPUs];
  float *d_accx[nGPUs], *d_accy[nGPUs], *d_accz[nGPUs], *d_mass[nGPUs];

  float *d_vx[nGPUs], *d_vy[nGPUs], *d_vz[nGPUs];
  float *d_h[nGPUs], *d_rho[nGPUs];

  float *d_P[nGPUs], *d_csnd[nGPUs], *d_divV[nGPUs], *d_curlV[nGPUs];
  float *d_accx_sph[nGPUs], *d_accy_sph[nGPUs], *d_accz_sph[nGPUs];
  float *d_accx_tot[nGPUs], *d_accy_tot[nGPUs], *d_accz_tot[nGPUs];
  float *d_abs_acc_g[nGPUs], *d_abs_acc_tot[nGPUs];
  float *d_v_sig[nGPUs], *d_dh_dt[nGPUs], *d_u[nGPUs], *d_dudt[nGPUs];
  float *d_utprevious[nGPUs];
  float *d_Nngb_previous[nGPUs]; // Note that both are floats and not int! check smoothing func. to see why!
  float *d_dt_particles[nGPUs];

  for (int i = 0; i < nGPUs; i++)
  {

    cout << "i = " << i << endl;

    hipSetDevice(i);

    // Allocate device memory for full arrays
    hipMalloc(&d_Typ[i], N * sizeof(int));

    hipMalloc(&d_x[i], N * sizeof(float));
    hipMalloc(&d_y[i], N * sizeof(float));
    hipMalloc(&d_z[i], N * sizeof(float));

    hipMalloc(&d_vx[i], N * sizeof(float));
    hipMalloc(&d_vy[i], N * sizeof(float));
    hipMalloc(&d_vz[i], N * sizeof(float));

    hipMalloc(&d_accx[i], N * sizeof(float));
    hipMalloc(&d_accy[i], N * sizeof(float));
    hipMalloc(&d_accz[i], N * sizeof(float));

    hipMalloc(&d_eps[i], N * sizeof(float));
    hipMalloc(&d_mass[i], N * sizeof(float));

    hipMalloc(&d_h[i], N * sizeof(float));
    hipMalloc(&d_rho[i], N * sizeof(float));
    hipMalloc(&d_P[i], N * sizeof(float));
    hipMalloc(&d_csnd[i], N * sizeof(float));

    hipMalloc(&d_divV[i], N * sizeof(float));
    hipMalloc(&d_curlV[i], N * sizeof(float));

    hipMalloc(&d_accx_sph[i], N * sizeof(float));
    hipMalloc(&d_accy_sph[i], N * sizeof(float));
    hipMalloc(&d_accz_sph[i], N * sizeof(float));

    hipMalloc(&d_accx_tot[i], N * sizeof(float));
    hipMalloc(&d_accy_tot[i], N * sizeof(float));
    hipMalloc(&d_accz_tot[i], N * sizeof(float));

    hipMalloc(&d_abs_acc_g[i], N * sizeof(float));
    hipMalloc(&d_abs_acc_tot[i], N * sizeof(float));
    hipMalloc(&d_v_sig[i], N * sizeof(float));
    hipMalloc(&d_dh_dt[i], N * sizeof(float));
    hipMalloc(&d_dt_particles[i], N * sizeof(float));

    hipMalloc(&d_u[i], N * sizeof(float));
    hipMalloc(&d_dudt[i], N * sizeof(float));
    hipMalloc(&d_utprevious[i], N * sizeof(float));

    hipMalloc(&d_Nngb_previous[i], N * sizeof(float));

    // Copy full arrays to each device
    hipMemcpy(d_Typ[i], Typ, N * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_x[i], x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y[i], y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z[i], z, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_vx[i], vx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy[i], vy, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vz[i], vz, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_accx[i], accx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accy[i], accy, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accz[i], accz, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_eps[i], eps, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mass[i], mass, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_h[i], h, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_rho[i], rho, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_P[i], P, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csnd[i], csnd, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_divV[i], divV, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_curlV[i], curlV, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_accx_sph[i], accx_sph, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accy_sph[i], accy_sph, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accz_sph[i], accz_sph, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_accx_tot[i], accx_tot, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accy_tot[i], accy_tot, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accz_tot[i], accz_tot, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_abs_acc_g[i], abs_acc_g, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_abs_acc_tot[i], abs_acc_tot, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v_sig[i], v_sig, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dh_dt[i], dh_dt, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dt_particles[i], dt_particles, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_u[i], u, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dudt[i], dudt, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_utprevious[i], utprevious, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_Nngb_previous[i], Nngb_previous, N * sizeof(float), hipMemcpyHostToDevice);
  }

  int blockSize = 256;                            // number of threads in a block
  int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks in a grid

  const float visc_alpha = 1.0f;

  float t = 0.0f;

  float tEnd = 5.0f;
  float Nt = ceil(tEnd / dt) + 1;

  //------------------------------------------------------------------
  // Determining the beg and end index of the array for each GPU.

  const int N_per_GPU = N / nGPUs;
  const int remainder = N % nGPUs;

  int *beg, *end, *MLen; // MLen means Memory length which is end - beg!

  beg = new int[nGPUs];
  end = new int[nGPUs];
  MLen = new int[nGPUs];

  for (int rank = 0; rank < nGPUs; rank++)
  {
    if (rank < remainder)
    {
      beg[rank] = rank * (N_per_GPU + 1);
      end[rank] = beg[rank] + N_per_GPU + 1;
      MLen[rank] = end[rank] - beg[rank];
    }
    else
    {
      beg[rank] = rank * N_per_GPU + remainder;
      end[rank] = beg[rank] + N_per_GPU;
      MLen[rank] = end[rank] - beg[rank];
    }
  }
  //------------------------

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!! START of smoothing_h !!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    cout << "AA" << endl;

    // Run the kernel only on a portion of the arrays
    smoothing_h_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i], d_h[i],
                                              Ndown, Nup, coeff, Nngb_f, d_Nngb_previous[i], d_divV[i], dt, N);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Now d_h[i] for each GPU, i, contains the results for its own portion portion of the data!

  // Copy results to main GPU.
  int NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_h[0] + NN, 0, d_h[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // The main GPU now has the full results in d_accx[0], d_accy[0], d_accz[0]

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_h[i], i, d_h[0], 0, N * sizeof(float));
  }

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF smoothing_h !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!! START of getDensity !!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    getDensity_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i], d_mass[i],
                                             d_rho[i], d_h[i], N);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Now d_accx[i], d_accy[i], and d_accz[i] for each GPU i contain the results for their portion

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_rho[0] + NN, 0, d_rho[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // The main GPU now has the full results in d_rho[0]

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_rho[i], i, d_rho[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!! END of getDensity !!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!! START of acc_g !!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    acc_g_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                        d_eps[i], d_accx[i], d_accy[i], d_accz[i], d_mass[i],
                                        G, N);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Now d_accx[i], d_accy[i], and d_accz[i] for each GPU i contain the results for their portion

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_accx[0] + NN, 0, d_accx[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_accy[0] + NN, 0, d_accy[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_accz[0] + NN, 0, d_accz[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // The main GPU now has the full results in d_accx[0], d_accy[0], d_accz[0]

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_accx[i], i, d_accx[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_accy[i], i, d_accy[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_accz[i], i, d_accz[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF acc_g !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF getPressure !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    getPressure_Adiabatic_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_P[i], d_rho[i], d_u[i],
                                                        gammah);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_P[0] + NN, 0, d_P[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_P[i], i, d_P[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF getPressure !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF getCsound !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    getCsound_Adiabatic_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_csnd[i], d_u[i], gammah);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_csnd[0] + NN, 0, d_csnd[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_csnd[i], i, d_csnd[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF getCsound !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF div_curlV !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    div_curlVel_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_divV[i], d_curlV[i],
                                              d_x[i], d_y[i], d_z[i],
                                              d_vx[i], d_vy[i], d_vz[i],
                                              d_rho[i], d_mass[i], d_h[i], N);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_divV[0] + NN, 0, d_divV[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_curlV[0] + NN, 0, d_curlV[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_divV[i], i, d_divV[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_curlV[i], i, d_curlV[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF div_curlV !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF acc_sph !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    acc_sph_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                          d_vx[i], d_vy[i], d_vz[i], d_h[i], d_csnd[i], d_rho[i],
                                          d_divV[i], d_curlV[i], d_mass[i], d_P[i],
                                          d_accx_sph[i], d_accy_sph[i], d_accz_sph[i], visc_alpha, N);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_accx_sph[0] + NN, 0, d_accx_sph[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_accy_sph[0] + NN, 0, d_accy_sph[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_accz_sph[0] + NN, 0, d_accz_sph[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_accx_sph[i], i, d_accx_sph[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_accy_sph[i], i, d_accy_sph[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_accz_sph[i], i, d_accz_sph[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF acc_sph !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF acc_tot !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    acc_g_sph_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i],
                                            d_accx_tot[i], d_accy_tot[i], d_accz_tot[i],
                                            d_accx[i], d_accy[i], d_accz[i],
                                            d_accx_sph[i], d_accy_sph[i], d_accz_sph[i]);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_accx_tot[0] + NN, 0, d_accx_tot[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_accy_tot[0] + NN, 0, d_accy_tot[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_accz_tot[0] + NN, 0, d_accz_tot[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_accx_tot[i], i, d_accx_tot[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_accy_tot[i], i, d_accy_tot[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_accz_tot[i], i, d_accz_tot[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF acc_tot !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF du_dt !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    get_dU_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                         d_vx[i], d_vy[i], d_vz[i], d_h[i], d_csnd[i], d_rho[i],
                                         d_divV[i], d_curlV[i], d_mass[i], d_P[i], d_dudt[i],
                                         visc_alpha, N);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_dudt[0] + NN, 0, d_dudt[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_dudt[i], i, d_dudt[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF du_dt !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! START OF u evolution !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);

    // Run the kernel only on a portion of the arrays
    u_updater_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_u[i], d_dudt[i],
                                            d_utprevious[i], dt);
  }

  // Synchronize to make sure computation is done before proceeding
  for (int i = 0; i < nGPUs; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  // Copy results to main GPU.
  NN = MLen[0];
  for (int i = 1; i < nGPUs; i++)
  {

    hipMemcpyPeer(d_u[0] + NN, 0, d_u[i] + NN, i, MLen[i] * sizeof(float));
    hipMemcpyPeer(d_utprevious[0] + NN, 0, d_utprevious[i] + NN, i, MLen[i] * sizeof(float));

    NN = NN + MLen[i];
  }

  // Copy results back to each device for next iteration
  for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
  {
    hipMemcpyPeer(d_u[i], i, d_u[0], 0, N * sizeof(float));
    hipMemcpyPeer(d_utprevious[i], i, d_utprevious[0], 0, N * sizeof(float));
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!! END OF u evolution !!!!!!!!!!!!!!!!!!!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  hipSetDevice(0); // Set the first GPU as active

  float leftover_mass = 0.0f;
  float *d_leftover_mass;
  hipMalloc((void **)&d_leftover_mass, sizeof(float));
  hipMemcpy(d_leftover_mass, &leftover_mass, sizeof(float), hipMemcpyHostToDevice);

  // **************************************************************
  // **************************************************************
  // *********************** MAIN LOOP ****************************
  // **************************************************************
  // **************************************************************

  int counter = 0; // This is used to save fewer output files, e.g. 1 snap-shot per 2 time-step!

  while (t < tEnd)
  {

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF velocity evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      v_evolve_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_vx[i], d_vy[i], d_vz[i],
                                             d_accx_tot[i], d_accy_tot[i], d_accz_tot[i], dt);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_vx[0] + NN, 0, d_vx[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_vy[0] + NN, 0, d_vy[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_vz[0] + NN, 0, d_vz[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_vx[i], i, d_vx[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_vy[i], i, d_vy[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_vz[i], i, d_vz[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF velocity evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF position evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      r_evolve_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                             d_vx[i], d_vy[i], d_vz[i], dt);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_x[0] + NN, 0, d_x[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_y[0] + NN, 0, d_y[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_z[0] + NN, 0, d_z[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_x[i], i, d_x[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_y[i], i, d_y[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_z[i], i, d_z[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF position evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!! START of smoothing_h !!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      smoothing_h_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i], d_h[i],
                                                Ndown, Nup, coeff, Nngb_f, d_Nngb_previous[i], d_divV[i], dt, N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Now d_h[i] for each GPU, i, contains the results for its own portion portion of the data!

    // Copy results to main GPU.
    int NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_h[0] + NN, 0, d_h[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // The main GPU now has the full results in d_accx[0], d_accy[0], d_accz[0]

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_h[i], i, d_h[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF smoothing_h !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!! START OF (Set eps of Gas equal to h) !!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      set_eps_of_gas_to_h_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_eps[i], d_h[i]);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Now d_h[i] for each GPU, i, contains the results for its own portion portion of the data!

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_eps[0] + NN, 0, d_eps[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // The main GPU now has the full results in d_accx[0], d_accy[0], d_accz[0]

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_eps[i], i, d_eps[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!! END OF (Set eps of Gas equal to h) !!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!! START of getDensity !!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      getDensity_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i], d_mass[i],
                                               d_rho[i], d_h[i], N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Now d_accx[i], d_accy[i], and d_accz[i] for each GPU i contain the results for their portion

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_rho[0] + NN, 0, d_rho[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // The main GPU now has the full results in d_rho[0]

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_rho[i], i, d_rho[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!! END of getDensity !!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!! START of acc_g !!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      acc_g_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                          d_eps[i], d_accx[i], d_accy[i], d_accz[i], d_mass[i],
                                          G, N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Now d_accx[i], d_accy[i], and d_accz[i] for each GPU i contain the results for their portion

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_accx[0] + NN, 0, d_accx[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accy[0] + NN, 0, d_accy[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accz[0] + NN, 0, d_accz[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // The main GPU now has the full results in d_accx[0], d_accy[0], d_accz[0]

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_accx[i], i, d_accx[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accy[i], i, d_accy[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accz[i], i, d_accz[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF acc_g !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF getPressure !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      getPressure_Adiabatic_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_P[i], d_rho[i], d_u[i],
                                                          gammah);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_P[0] + NN, 0, d_P[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_P[i], i, d_P[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF getPressure !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF getCsound !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      getCsound_Adiabatic_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_csnd[i], d_u[i], gammah);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_csnd[0] + NN, 0, d_csnd[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_csnd[i], i, d_csnd[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF getCsound !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF div_curlV !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      div_curlVel_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_divV[i], d_curlV[i],
                                                d_x[i], d_y[i], d_z[i],
                                                d_vx[i], d_vy[i], d_vz[i],
                                                d_rho[i], d_mass[i], d_h[i], N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_divV[0] + NN, 0, d_divV[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_curlV[0] + NN, 0, d_curlV[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_divV[i], i, d_divV[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_curlV[i], i, d_curlV[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF div_curlV !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF acc_sph !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      acc_sph_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                            d_vx[i], d_vy[i], d_vz[i], d_h[i], d_csnd[i], d_rho[i],
                                            d_divV[i], d_curlV[i], d_mass[i], d_P[i],
                                            d_accx_sph[i], d_accy_sph[i], d_accz_sph[i], visc_alpha, N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_accx_sph[0] + NN, 0, d_accx_sph[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accy_sph[0] + NN, 0, d_accy_sph[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accz_sph[0] + NN, 0, d_accz_sph[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_accx_sph[i], i, d_accx_sph[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accy_sph[i], i, d_accy_sph[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accz_sph[i], i, d_accz_sph[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF acc_sph !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF acc_tot !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      acc_g_sph_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i],
                                              d_accx_tot[i], d_accy_tot[i], d_accz_tot[i],
                                              d_accx[i], d_accy[i], d_accz[i],
                                              d_accx_sph[i], d_accy_sph[i], d_accz_sph[i]);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_accx_tot[0] + NN, 0, d_accx_tot[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accy_tot[0] + NN, 0, d_accy_tot[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accz_tot[0] + NN, 0, d_accz_tot[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_accx_tot[i], i, d_accx_tot[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accy_tot[i], i, d_accy_tot[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accz_tot[i], i, d_accz_tot[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF acc_tot !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //! START OF Isothermal Gravity (Richings et al - 2018) !!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      galaxy_isothermal_potential_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i],
                                                                d_x[i], d_y[i], d_z[i], d_accx_tot[i],
                                                                d_accy_tot[i], d_accz_tot[i], sigma, G);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_accx_tot[0] + NN, 0, d_accx_tot[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accy_tot[0] + NN, 0, d_accy_tot[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_accz_tot[0] + NN, 0, d_accz_tot[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_accx_tot[i], i, d_accx_tot[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accy_tot[i], i, d_accy_tot[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_accz_tot[i], i, d_accz_tot[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //! END OF Isothermal Gravity (Richings et al - 2018) !!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF velocity evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      v_evolve_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_vx[i], d_vy[i], d_vz[i],
                                             d_accx_tot[i], d_accy_tot[i], d_accz_tot[i], dt);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_vx[0] + NN, 0, d_vx[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_vy[0] + NN, 0, d_vy[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_vz[0] + NN, 0, d_vz[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_vx[i], i, d_vx[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_vy[i], i, d_vy[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_vz[i], i, d_vz[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF velocity evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF du_dt !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      get_dU_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_x[i], d_y[i], d_z[i],
                                           d_vx[i], d_vy[i], d_vz[i], d_h[i], d_csnd[i], d_rho[i],
                                           d_divV[i], d_curlV[i], d_mass[i], d_P[i], d_dudt[i],
                                           visc_alpha, N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_dudt[0] + NN, 0, d_dudt[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_dudt[i], i, d_dudt[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF du_dt !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! START OF u evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      u_updater_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i], d_u[i], d_dudt[i],
                                              d_utprevious[i], dt);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_u[0] + NN, 0, d_u[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_utprevious[0] + NN, 0, d_utprevious[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_u[i], i, d_u[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_utprevious[i], i, d_utprevious[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!! END OF u evolution !!!!!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    //------------ SAVING SNAP-SHOTS ------------
    if (!(counter % 100))
    {
      hipMemcpy(Typ, d_Typ[0], N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(x, d_x[0], N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y[0], N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(z, d_z[0], N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(vx, d_vx[0], N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vy, d_vy[0], N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vz, d_vz[0], N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(rho, d_rho[0], N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(h, d_h[0], N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(u, d_u[0], N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(mass, d_mass[0], N * sizeof(float), hipMemcpyDeviceToHost);

      // Specify the output file name
      std::string filename = "./Outputs/G-" + to_string(t * 1) + ".bin";
      // Save the arrays to binary format
      saveArraysToBinary(filename, x, y, z, vx, vy, vz, rho, h, u, mass, Typ, N);
    }

    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!! START OF Updating Time-step dt !!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);

      // Run the kernel only on a portion of the arrays
      dt_array_indiv_dt_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], d_Typ[i],
                                                      d_x[i], d_y[i], d_z[i],
                                                      d_vx[i], d_vy[i], d_vz[i],
                                                      d_accx[i], d_accy[i], d_accz[i],
                                                      d_accx_tot[i], d_accy_tot[i], d_accz_tot[i],
                                                      d_h[i], d_csnd[i], d_dt_particles[i],
                                                      d_abs_acc_g[i], d_abs_acc_tot[i],
                                                      d_divV[i], d_dh_dt[i], C_CFL,
                                                      visc_alpha, d_eps[i], N);
    }

    // Synchronize to make sure computation is done before proceeding
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Copy results to main GPU.
    NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(d_dt_particles[0] + NN, 0, d_dt_particles[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(d_dh_dt[0] + NN, 0, d_dh_dt[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(d_dt_particles[i], i, d_dt_particles[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_dh_dt[i], i, d_dh_dt[0], 0, N * sizeof(float));
    }
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //!!!!!!!! END OF Updating Time-step dt !!!!!!!!!!!!!!!!
    //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    t += dt;

    // dt = min_finder(d_Typ[0], d_dt_particles[0], N);

    hipSetDevice(0); // One GPU is enough for this function!

    h_min = min_finder(d_Typ[0], d_h[0], N);
    h_max = max_finder(d_Typ[0], d_h[0], N);
    h_mean = 0.5f * (h_min + h_max);
    cout << "KK" << endl;

    // Generate a seed using the high resolution clock
    auto now = std::chrono::high_resolution_clock::now();
    auto nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
    unsigned long long seed = static_cast<unsigned long long>(nanos);

    outflow_injector<<<gridSize, blockSize>>>(d_Typ[0], d_x[0], d_y[0], d_z[0],
                                              d_vx[0], d_vy[0], d_vz[0],
                                              d_h[0], d_eps[0], d_mass[0],
                                              Nngb_f, d_Nngb_previous[0],
                                              d_u[0], M_dot_in, v_in,
                                              m_sph_high_res, u_for_10K_Temp,
                                              h_mean, d_leftover_mass, dt, N,
                                              seed);
    hipDeviceSynchronize();

    // UPDATE EVERY THING!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    // Since new particles are added by GPU 0, therefore all arrays must be updated in all other GPUs!
    // Copy results back to each device for next iteration
    for (int i = 1; i < nGPUs; i++) // Note that GPU 0 already has the data!
    {
      hipMemcpyPeer(d_Typ[i], i, d_Typ[0], 0, N * sizeof(int));

      hipMemcpyPeer(d_x[i], i, d_x[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_y[i], i, d_y[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_z[i], i, d_z[0], 0, N * sizeof(float));

      hipMemcpyPeer(d_vx[i], i, d_vx[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_vy[i], i, d_vy[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_vz[i], i, d_vz[0], 0, N * sizeof(float));

      hipMemcpyPeer(d_h[i], i, d_h[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_eps[i], i, d_eps[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_mass[i], i, d_mass[0], 0, N * sizeof(float));

      hipMemcpyPeer(d_u[i], i, d_u[0], 0, N * sizeof(float));
      hipMemcpyPeer(d_Nngb_previous[i], i, d_Nngb_previous[0], 0, N * sizeof(float));
    }

    if (!(counter % 1))
    {
      cout << "Adopted dt = " << dt << endl;
      cout << "current t = " << t << endl;
      cout << "*****************************" << endl;
      cout << endl;
    }

    counter++;

  } // The end of WHILE loop

} // The end of main().
