#include "hip/hip_runtime.h"
//%%writefile test.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <random>
#include <tuple>
#include "multi_libs.h"
#include "gpu_helpers.h"

using namespace std;

// Function to save results
void saveArraysToBinary(const std::string &filename, float *accx, float *accy, float *accz, size_t N)
{
  std::ofstream out(filename, std::ios::binary);
  out.write(reinterpret_cast<const char *>(accx), N * sizeof(float));
  out.write(reinterpret_cast<const char *>(accy), N * sizeof(float));
  out.write(reinterpret_cast<const char *>(accz), N * sizeof(float));
  out.close();
}

int main()
{

  // Reading the params.txt file
  std::string filename;
  int N;
  float G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma;

  readParams(filename, N, G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma);

  std::cout << "filename: " << filename << "\n";
  std::cout << "N: " << N << "\n";
  std::cout << "G: " << G << "\n";
  std::cout << "L_AGN_code_unit: " << L_AGN_code_unit << "\n";
  std::cout << "M_dot_in_code_unit: " << M_dot_in << "\n";
  std::cout << "vin_in_code_unit: " << v_in << "\n";
  std::cout << "u_for_10K_Temp: " << u_for_10K_Temp << "\n";
  std::cout << "m_sph_high_res: " << m_sph_high_res << "\n";
  std::cout << "sigma: " << sigma << "\n";

  // Reading the IC file
  auto data = readVectorsFromFile(N, filename);

  std::vector<int> &Typvec = std::get<0>(data);
  std::vector<float> &xvec = std::get<1>(data);
  std::vector<float> &yvec = std::get<2>(data);
  std::vector<float> &zvec = std::get<3>(data);
  std::vector<float> &vxvec = std::get<4>(data);
  std::vector<float> &vyvec = std::get<5>(data);
  std::vector<float> &vzvec = std::get<6>(data);
  std::vector<float> &massvec = std::get<7>(data);
  std::vector<float> &hvec = std::get<8>(data);
  std::vector<float> &epsvec = std::get<9>(data);
  std::vector<float> &uvec = std::get<10>(data);

  // Allocate and initialize host memory for full arrays
  int *Typ = new int[N];
  float *x = new float[N];
  float *y = new float[N];
  float *z = new float[N];
  float *eps = new float[N];
  float *mass = new float[N];

  float *accx = new float[N];
  float *accy = new float[N];
  float *accz = new float[N];

  for (int i = 0; i < N; i++)
  {
    Typ[i] = Typvec[i];
    x[i] = xvec[i];
    y[i] = yvec[i];
    z[i] = zvec[i];
    eps[i] = epsvec[i];
    mass[i] = massvec[i];
    accx[i] = 0.0f;
    accy[i] = 0.0f;
    accz[i] = 0.0f;
  }

  int devCount;
  hipGetDeviceCount(&devCount);

  // Pointers for device memory
  int *dev_Typ[devCount];
  float *dev_x[devCount], *dev_y[devCount], *dev_z[devCount], *dev_eps[devCount];
  float *dev_accx[devCount], *dev_accy[devCount], *dev_accz[devCount], *dev_mass[devCount];

  for (int i = 0; i < devCount; i++)
  {
    hipSetDevice(i);

    // Allocate device memory for full arrays
    hipMalloc(&dev_Typ[i], N * sizeof(int));
    hipMalloc(&dev_x[i], N * sizeof(float));
    hipMalloc(&dev_y[i], N * sizeof(float));
    hipMalloc(&dev_z[i], N * sizeof(float));
    hipMalloc(&dev_eps[i], N * sizeof(float));
    hipMalloc(&dev_accx[i], N * sizeof(float));
    hipMalloc(&dev_accy[i], N * sizeof(float));
    hipMalloc(&dev_accz[i], N * sizeof(float));
    hipMalloc(&dev_mass[i], N * sizeof(float));

    // Copy full arrays to each device
    hipMemcpy(dev_Typ[i], Typ, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_x[i], x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y[i], y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_z[i], z, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_eps[i], eps, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_mass[i], mass, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_accx[i], accx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_accy[i], accy, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_accz[i], accz, N * sizeof(float), hipMemcpyHostToDevice);
  }

  int blockSize = 256;                            // number of threads in a block
  int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks in a grid

  //------------------------------------------------------------------
  // Determining the beg and end index of the array for each GPU.

  int nGPUs = 8; // devCount;

  cout << "nGPUs = " << nGPUs << endl;

  const int N_per_GPU = N / nGPUs;
  const int remainder = N % nGPUs;

  int *beg, *end, *MLen; // MLen means Memory length which is end - beg!

  beg = new int[nGPUs];
  end = new int[nGPUs];
  MLen = new int[nGPUs];

  for (int rank = 0; rank < nGPUs; rank++)
  {
    if (rank < remainder)
    {
      beg[rank] = rank * (N_per_GPU + 1);
      end[rank] = beg[rank] + N_per_GPU + 1;
      MLen[rank] = end[rank] - beg[rank];
    }
    else
    {
      beg[rank] = rank * N_per_GPU + remainder;
      end[rank] = beg[rank] + N_per_GPU;
      MLen[rank] = end[rank] - beg[rank];
    }
  }
  //------------------------

  int iterations = 1;

  for (int iter = 0; iter < iterations; iter++)
  {

    acc_g_on_multi_gpus(nGPUs, gridSize, blockSize, beg, end, dev_Typ, dev_x, dev_y, dev_z,
                        dev_eps, dev_accx, dev_accy, dev_accz, dev_mass,
                        G, N, MLen, devCount);

  } // ===> The end of iteration!

  // Save to a file
  // Copy data from GPU to host
  hipMemcpy(accx, dev_accx[0], N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(accy, dev_accy[0], N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(accz, dev_accz[0], N * sizeof(float), hipMemcpyDeviceToHost);
  // Save the arrays to binary format
  filename = "outMultiXX.bin";
  saveArraysToBinary(filename, accx, accy, accz, N);

} // The end of main().
