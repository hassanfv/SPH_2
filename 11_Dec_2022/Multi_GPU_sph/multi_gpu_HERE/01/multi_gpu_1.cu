#include "hip/hip_runtime.h"
//%%writefile test.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <random>
#include <tuple>
#include "multi_libs.h"

using namespace std;

// Function to save results
void saveArraysToBinary(const std::string &filename, float *accx, float *accy, float *accz, size_t N)
{
  std::ofstream out(filename, std::ios::binary);
  out.write(reinterpret_cast<const char *>(accx), N * sizeof(float));
  out.write(reinterpret_cast<const char *>(accy), N * sizeof(float));
  out.write(reinterpret_cast<const char *>(accz), N * sizeof(float));
  out.close();
}

int main()
{

  // Reading the params.txt file
  std::string filename;
  int N;
  float G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma;

  readParams(filename, N, G, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma);

  std::cout << "filename: " << filename << "\n";
  std::cout << "N: " << N << "\n";
  std::cout << "G: " << G << "\n";
  std::cout << "L_AGN_code_unit: " << L_AGN_code_unit << "\n";
  std::cout << "M_dot_in_code_unit: " << M_dot_in << "\n";
  std::cout << "vin_in_code_unit: " << v_in << "\n";
  std::cout << "u_for_10K_Temp: " << u_for_10K_Temp << "\n";
  std::cout << "m_sph_high_res: " << m_sph_high_res << "\n";
  std::cout << "sigma: " << sigma << "\n";

  // Reading the IC file
  auto data = readVectorsFromFile(N, filename);

  std::vector<int> &Typvec = std::get<0>(data);
  std::vector<float> &xvec = std::get<1>(data);
  std::vector<float> &yvec = std::get<2>(data);
  std::vector<float> &zvec = std::get<3>(data);
  std::vector<float> &vxvec = std::get<4>(data);
  std::vector<float> &vyvec = std::get<5>(data);
  std::vector<float> &vzvec = std::get<6>(data);
  std::vector<float> &massvec = std::get<7>(data);
  std::vector<float> &hvec = std::get<8>(data);
  std::vector<float> &epsvec = std::get<9>(data);
  std::vector<float> &uvec = std::get<10>(data);

  // Allocate and initialize host memory for full arrays
  int *Typ = new int[N];
  float *x = new float[N];
  float *y = new float[N];
  float *z = new float[N];
  float *eps = new float[N];
  float *mass = new float[N];

  float *accx = new float[N];
  float *accy = new float[N];
  float *accz = new float[N];

  for (int i = 0; i < N; i++)
  {
    Typ[i] = Typvec[i];
    x[i] = xvec[i];
    y[i] = yvec[i];
    z[i] = zvec[i];
    eps[i] = epsvec[i];
    mass[i] = massvec[i];
    accx[i] = 0.0f;
    accy[i] = 0.0f;
    accz[i] = 0.0f;
  }

  int devCount;
  hipGetDeviceCount(&devCount);

  // Pointers for device memory
  int *dev_Typ[devCount];
  float *dev_x[devCount], *dev_y[devCount], *dev_z[devCount], *dev_eps[devCount];
  float *dev_accx[devCount], *dev_accy[devCount], *dev_accz[devCount], *dev_mass[devCount];

  for (int i = 0; i < devCount; i++)
  {
    hipSetDevice(i);

    // Allocate device memory for full arrays
    hipMalloc(&dev_Typ[i], N * sizeof(int));
    hipMalloc(&dev_x[i], N * sizeof(float));
    hipMalloc(&dev_y[i], N * sizeof(float));
    hipMalloc(&dev_z[i], N * sizeof(float));
    hipMalloc(&dev_eps[i], N * sizeof(float));
    hipMalloc(&dev_accx[i], N * sizeof(float));
    hipMalloc(&dev_accy[i], N * sizeof(float));
    hipMalloc(&dev_accz[i], N * sizeof(float));
    hipMalloc(&dev_mass[i], N * sizeof(float));

    // Copy full arrays to each device
    hipMemcpy(dev_Typ[i], Typ, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_x[i], x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y[i], y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_z[i], z, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_eps[i], eps, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_mass[i], mass, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_accx[i], accx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_accy[i], accy, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_accz[i], accz, N * sizeof(float), hipMemcpyHostToDevice);
  }

  int blockSize = 256;                            // number of threads in a block
  int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks in a grid

  //------------------------------------------------------------------
  // Determining the beg and end index of the array for each GPU.

  int nGPUs = 4; // devCount;

  cout << "nGPUs = " << nGPUs << endl;

  const int N_per_GPU = N / nGPUs;
  const int remainder = N % nGPUs;

  int *beg, *end, *MLen; // MLen means Memory length which is end - beg!

  beg = new int[nGPUs];
  end = new int[nGPUs];
  MLen = new int[nGPUs];

  for (int rank = 0; rank < nGPUs; rank++)
  {
    if (rank < remainder)
    {
      beg[rank] = rank * (N_per_GPU + 1);
      end[rank] = beg[rank] + N_per_GPU + 1;
      MLen[rank] = end[rank] - beg[rank];
    }
    else
    {
      beg[rank] = rank * N_per_GPU + remainder;
      end[rank] = beg[rank] + N_per_GPU;
      MLen[rank] = end[rank] - beg[rank];
    }
  }
  //------------------------

  int iterations = 10;

  for (int iter = 0; iter < iterations; iter++)
  {

    //================ acc_g ====================
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      acc_g_mgpu<<<gridSize, blockSize>>>(beg[i], end[i], dev_Typ[i], dev_x[i], dev_y[i], dev_z[i],
                                          dev_eps[i], dev_accx[i], dev_accy[i], dev_accz[i], dev_mass[i],
                                          G, N);
    }

    // Synchronize
    for (int i = 0; i < nGPUs; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    // Now dev_accx[i], dev_accy[i], dev_accz[i] for each GPU, i, contain the results of their own portion of the data.

    // Copy the results to main GPU.
    int NN = MLen[0];
    for (int i = 1; i < nGPUs; i++)
    {

      hipMemcpyPeer(dev_accx[0] + NN, 0, dev_accx[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(dev_accy[0] + NN, 0, dev_accy[i] + NN, i, MLen[i] * sizeof(float));
      hipMemcpyPeer(dev_accz[0] + NN, 0, dev_accz[i] + NN, i, MLen[i] * sizeof(float));

      NN = NN + MLen[i];
    }

    // The main GPU now has the full results in dev_accx[0], dev_accy[0], dev_accz[0]

    // Copy the full results back to each device for next iteration (i.e. time-step).
    for (int i = 1; i < devCount; i++) // Note that GPU 0 already hass the data!
    {
      hipMemcpyPeer(dev_accx[i], i, dev_accx[0], 0, N * sizeof(float));
      hipMemcpyPeer(dev_accy[i], i, dev_accy[0], 0, N * sizeof(float));
      hipMemcpyPeer(dev_accz[i], i, dev_accz[0], 0, N * sizeof(float));
    }

  } // ===> The end of iteration!

} // The end of main().
