#include "hip/hip_runtime.h"
//%%writefile test.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <random>
#include <tuple>
#include "hfvCppLibs_v2.h"


// Added the reading of the params.txt file and updated the IC reading file section and function. (22 May 2023).

using namespace std;

int main()
{

  float dt = 1e-4; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! This is only the first time step !!

  const int Nngb_f = 64.0f; // used in smoothing func.
  const int Nngb = 64;
  const int Ndown = Nngb - 5;
  const int Nup = Nngb + 5;
  const float coeff = 0.001f; // used for smoothing length.


  // Reading the params.txt file
  std::string filename;
  int N;
  float G, L_AGN_code_unit, M_dot_in_code_unit, vin_in_code_unit, u_for_10K_Temp, m_sph_high_res;

  readParams(filename, N, G, L_AGN_code_unit, M_dot_in_code_unit, vin_in_code_unit, u_for_10K_Temp, m_sph_high_res);

  std::cout << "filename: " << filename << "\n";
  std::cout << "N: " << N << "\n";
  std::cout << "G: " << G << "\n";
  std::cout << "L_AGN_code_unit: " << L_AGN_code_unit << "\n";
  std::cout << "M_dot_in_code_unit: " << M_dot_in_code_unit << "\n";
  std::cout << "vin_in_code_unit: " << vin_in_code_unit << "\n";
  std::cout << "u_for_10K_Temp: " << u_for_10K_Temp << "\n";
  std::cout << "m_sph_high_res: " << m_sph_high_res << "\n";

  // int N = 1472;
  //std::string filename = "Evrard_GPU_IC_1k.bin";
  // Reading the IC file
  auto data = readVectorsFromFile(N, filename);

  std::vector<int> &Typvec = std::get<0>(data);
  std::vector<float> &xvec = std::get<1>(data);
  std::vector<float> &yvec = std::get<2>(data);
  std::vector<float> &zvec = std::get<3>(data);
  std::vector<float> &vxvec = std::get<4>(data);
  std::vector<float> &vyvec = std::get<5>(data);
  std::vector<float> &vzvec = std::get<6>(data);
  std::vector<float> &massvec = std::get<7>(data);
  std::vector<float> &hvec = std::get<8>(data);
  std::vector<float> &epsvec = std::get<9>(data);
  std::vector<float> &uvec = std::get<10>(data);

  // declaring the arrays.
  int *Typ, *d_Typ;
  float *x, *d_x, *y, *d_y, *z, *d_z, *vx, *d_vx, *vy, *d_vy, *vz, *d_vz;
  float *mass, *d_mass, *h, *d_h, *rho, *d_rho;
  float *accx, *accy, *accz, *d_accx, *d_accy, *d_accz, *eps, *d_eps;
  float *P, *d_P, *csnd, *d_csnd, *divV, *d_divV, *curlV, *d_curlV;
  float *accx_sph, *accy_sph, *accz_sph, *d_accx_sph, *d_accy_sph, *d_accz_sph;
  float *accx_tot, *accy_tot, *accz_tot, *d_accx_tot, *d_accy_tot, *d_accz_tot;
  float *abs_acc_g, *abs_acc_tot, *v_sig, *dh_dt, *d_abs_acc_g, *d_abs_acc_tot;
  float *d_v_sig, *d_dh_dt, *u, *dudt, *d_u, *d_dudt, *utprevious;
  float *d_utprevious;
  float *Nngb_previous, *d_Nngb_previous; // Note that both are floats and not int! check smoothing func. to see why!
  float *dt_particles, *d_dt_particles;

  float gammah = 5.0f / 3.0f;

  Typ = new int[N];

  x = new float[N];
  y = new float[N];
  z = new float[N];

  vx = new float[N];
  vy = new float[N];
  vz = new float[N];

  accx = new float[N];
  accy = new float[N];
  accz = new float[N];

  mass = new float[N];
  h = new float[N];
  rho = new float[N];
  eps = new float[N];
  P = new float[N];
  csnd = new float[N];

  divV = new float[N];
  curlV = new float[N];

  accx_sph = new float[N];
  accy_sph = new float[N];
  accz_sph = new float[N];

  accx_tot = new float[N];
  accy_tot = new float[N];
  accz_tot = new float[N];

  abs_acc_g = new float[N];
  abs_acc_tot = new float[N];
  v_sig = new float[N];
  dh_dt = new float[N];
  dt_particles = new float[N];

  u = new float[N];
  dudt = new float[N];
  utprevious = new float[N];

  Nngb_previous = new float[N];

  hipMalloc(&d_Typ, N * sizeof(int));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  hipMalloc(&d_vx, N * sizeof(float));
  hipMalloc(&d_vy, N * sizeof(float));
  hipMalloc(&d_vz, N * sizeof(float));

  hipMalloc(&d_accx, N * sizeof(float));
  hipMalloc(&d_accy, N * sizeof(float));
  hipMalloc(&d_accz, N * sizeof(float));

  hipMalloc(&d_mass, N * sizeof(float));
  hipMalloc(&d_h, N * sizeof(float));
  hipMalloc(&d_rho, N * sizeof(float));
  hipMalloc(&d_eps, N * sizeof(float));
  hipMalloc(&d_P, N * sizeof(float));
  hipMalloc(&d_csnd, N * sizeof(float));

  hipMalloc(&d_divV, N * sizeof(float));
  hipMalloc(&d_curlV, N * sizeof(float));

  hipMalloc(&d_accx_sph, N * sizeof(float));
  hipMalloc(&d_accy_sph, N * sizeof(float));
  hipMalloc(&d_accz_sph, N * sizeof(float));

  hipMalloc(&d_accx_tot, N * sizeof(float));
  hipMalloc(&d_accy_tot, N * sizeof(float));
  hipMalloc(&d_accz_tot, N * sizeof(float));

  hipMalloc(&d_abs_acc_g, N * sizeof(float));
  hipMalloc(&d_abs_acc_tot, N * sizeof(float));
  hipMalloc(&d_v_sig, N * sizeof(float));
  hipMalloc(&d_dh_dt, N * sizeof(float));
  hipMalloc(&d_dt_particles, N * sizeof(float));

  hipMalloc(&d_u, N * sizeof(float));
  hipMalloc(&d_dudt, N * sizeof(float));
  hipMalloc(&d_utprevious, N * sizeof(float));

  hipMalloc(&d_Nngb_previous, N * sizeof(float));

  // Initialize x, y, and z on the Host.
  for (int i = 0; i < N; i++)
  {
    Typ[i] = Typvec[i];

    x[i] = xvec[i];
    y[i] = yvec[i];
    z[i] = zvec[i];

    vx[i] = vxvec[i];
    vy[i] = vyvec[i];
    vz[i] = vzvec[i];

    mass[i] = massvec[i];
    eps[i] = epsvec[i];

    accx[i] = 0.0f;
    accy[i] = 0.0f;
    accz[i] = 0.0f;

    accx_tot[i] = 0.0f;
    accy_tot[i] = 0.0f;
    accz_tot[i] = 0.0f;

    abs_acc_g[i] = 0.0f;
    abs_acc_tot[i] = 0.0f;
    v_sig[i] = 0.0f;

    h[i] = hvec[i]; // place holder.
    rho[i] = 0.0f;  // place holder.
    P[i] = 0.0f;    // placeholder.
    csnd[i] = 0.0f; // placeholder.

    divV[i] = 0.0f;  // placeholder.
    curlV[i] = 0.0f; // placeholder.

    accx_sph[i] = 0.0f;
    accy_sph[i] = 0.0f;
    accz_sph[i] = 0.0f;

    dh_dt[i] = 0.0f;

    u[i] = uvec[i];
    dudt[i] = 0.0f;
    utprevious[i] = 0.0f;

    dt_particles[i] = 0.0f;

    if (Typ[i] == 0)
    {
    Nngb_previous[i] = Nngb_f;
    } else
    {
    Nngb_previous[i] = 0.0f;
    }
  }

  // Copy from Host to Device.
  hipMemcpy(d_Typ, Typ, N * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_vx, vx, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vy, vy, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vz, vz, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx, accx, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy, accy, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz, accz, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_mass, mass, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_h, h, N * sizeof(float), hipMemcpyHostToDevice);

  // hipMemcpy(d_hprevious, hprevious, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rho, rho, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_eps, eps, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_P, P, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_csnd, csnd, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_divV, divV, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_curlV, curlV, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx_sph, accx_sph, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy_sph, accy_sph, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz_sph, accz_sph, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx_tot, accx_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy_tot, accy_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz_tot, accz_tot, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_abs_acc_g, abs_acc_g, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_abs_acc_tot, abs_acc_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_sig, v_sig, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dh_dt, dh_dt, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dt_particles, dt_particles, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_u, u, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dudt, dudt, N * sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(d_uprevious, uprevious, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_utprevious, utprevious, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_Nngb_previous, Nngb_previous, N * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;                            // number of threads in a block
  int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks in a grid

  const float visc_alpha = 1.0f;

  float t;

  t = 0.0f;

  float tEnd = 5.0f;
  float Nt = ceil(tEnd / dt) + 1;

  //-----------------------------------------------
  //-------------- Smoothing Length ---------------
  //-----------------------------------------------
  smoothing_h<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_h,
                                       N, Ndown, Nup, coeff,
                                       Nngb_f, d_Nngb_previous, d_divV, dt);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- getDensity ------------------
  //-----------------------------------------------
  getDensity<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_mass,
                                      d_rho, d_h, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------ getAcc_g -------------------
  //-----------------------------------------------
  acc_g<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_eps, d_accx, d_accy, d_accz,
                                 d_mass, G, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //---------------- getPressure ------------------
  //-----------------------------------------------
  getPressure_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_P, d_rho, d_u, gammah, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- getCsound -------------------
  //-----------------------------------------------
  getCsound_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_csnd, d_u, gammah, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- div_curlV -------------------
  //-----------------------------------------------
  div_curlVel<<<gridSize, blockSize>>>(d_Typ, d_divV, d_curlV, d_x, d_y, d_z, d_vx, d_vy, d_vz,
                                       d_rho, d_mass, d_h, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------ acc_sph --------------------
  //-----------------------------------------------
  acc_sph<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                   d_divV, d_curlV, d_mass, d_P, d_accx_sph, d_accy_sph,
                                   d_accz_sph, visc_alpha, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------ acc_tot --------------------
  //-----------------------------------------------
  acc_g_sph<<<gridSize, blockSize>>>(d_Typ, d_accx_tot, d_accy_tot, d_accz_tot,
                                     d_accx, d_accy, d_accz,
                                     d_accx_sph, d_accy_sph, d_accz_sph,
                                     N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------- du_dt ---------------------
  //-----------------------------------------------

  get_dU<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                  d_divV, d_curlV, d_mass, d_P, d_dudt,
                                  visc_alpha, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //---------------- u evolution ------------------
  //-----------------------------------------------

  u_updater<<<gridSize, blockSize>>>(d_Typ, d_u, d_dudt, d_utprevious, dt, N);
  hipDeviceSynchronize();

  const float C_CFL = 0.25;

  // **************************************************************
  // *********************** MAIN LOOP ****************************
  // **************************************************************

  int counter = 0; // This is used to save fewer output files, e.g. 1 snap-shot per 2 time-step!

  while (t < tEnd)
  {

    auto begin = std::chrono::high_resolution_clock::now();

    //****************** velocity evolution *******************
    v_evolve<<<gridSize, blockSize>>>(d_Typ, d_vx, d_vy, d_vz, d_accx_tot, d_accy_tot,
                                      d_accz_tot, dt, N);
    hipDeviceSynchronize();

    //****************** position evolution (BH fixed at [0, 0, 0]) *******************

    r_evolve<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, dt, N);
    hipDeviceSynchronize();

    //****************** Smoothing Length *********************

    smoothing_h<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_h,
                                         N, Ndown, Nup, coeff,
                                         Nngb_f, d_Nngb_previous, d_divV, dt);
    hipDeviceSynchronize();

    //****************** Set eps of Gas equal to h ******************

    set_eps_of_gas_to_h<<<gridSize, blockSize>>>(d_Typ, d_eps, d_h, N);
    hipDeviceSynchronize();

    //****************** getDensity ***********************
    getDensity<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_mass,
                                        d_rho, d_h, N);
    hipDeviceSynchronize();

    //****************** getAcc_gX *************************
    auto T_acc_g = std::chrono::high_resolution_clock::now();
    acc_g<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_eps, d_accx, d_accy, d_accz,
                                   d_mass, G, N);
    hipDeviceSynchronize();
    auto end_acc_g = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_g = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_g - T_acc_g);
    cout << "T_acc_g = " << elapsed_acc_g.count() * 1e-9 << endl;

    //****************** getPressure **********************
    getPressure_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_P, d_rho, d_u, gammah, N);
    hipDeviceSynchronize();

    //****************** getCsound ************************
    getCsound_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_csnd, d_u, gammah, N);
    hipDeviceSynchronize();

    //****************** div_curlVX ************************
    auto T_divCurl = std::chrono::high_resolution_clock::now();
    div_curlVel<<<gridSize, blockSize>>>(d_Typ, d_divV, d_curlV, d_x, d_y, d_z, d_vx, d_vy, d_vz,
                                         d_rho, d_mass, d_h, N);
    hipDeviceSynchronize();
    auto end_divCurl = std::chrono::high_resolution_clock::now();
    auto elapsed_divCurl = std::chrono::duration_cast<std::chrono::nanoseconds>(end_divCurl - T_divCurl);
    cout << "T_divCurl = " << elapsed_divCurl.count() * 1e-9 << endl;

    //****************** acc_sphX **************************
    auto T_acc_sph = std::chrono::high_resolution_clock::now();
    acc_sph<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                     d_divV, d_curlV, d_mass, d_P, d_accx_sph, d_accy_sph,
                                     d_accz_sph, visc_alpha, N);
    hipDeviceSynchronize();
    auto end_acc_sph = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_sph = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_sph - T_acc_sph);
    cout << "T_acc_sph = " << elapsed_acc_sph.count() * 1e-9 << endl;

    //****************** acc_tot **************************
    auto T_acc_tot = std::chrono::high_resolution_clock::now();
    acc_g_sph<<<gridSize, blockSize>>>(d_Typ, d_accx_tot, d_accy_tot, d_accz_tot,
                                       d_accx, d_accy, d_accz,
                                       d_accx_sph, d_accy_sph, d_accz_sph,
                                       N);
    hipDeviceSynchronize();
    auto end_acc_tot = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_tot = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_tot - T_acc_tot);
    cout << "T_acc_tot = " << elapsed_acc_tot.count() * 1e-9 << endl;

    //****************** velocity evolution *******************
    v_evolve<<<gridSize, blockSize>>>(d_Typ, d_vx, d_vy, d_vz, d_accx_tot, d_accy_tot,
                                      d_accz_tot, dt, N);
    hipDeviceSynchronize();

    //******************** get_dUX (du_dt) *********************
    auto T_dU = std::chrono::high_resolution_clock::now();
    get_dU<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                    d_divV, d_curlV, d_mass, d_P, d_dudt,
                                    visc_alpha, N);
    hipDeviceSynchronize();
    auto end_dU = std::chrono::high_resolution_clock::now();
    auto elapsed_dU = std::chrono::duration_cast<std::chrono::nanoseconds>(end_dU - T_dU);
    cout << "T_dU = " << elapsed_dU.count() * 1e-9 << endl;

    //******************** u evolution *********************
    u_updater<<<gridSize, blockSize>>>(d_Typ, d_u, d_dudt, d_utprevious, dt, N);
    hipDeviceSynchronize();

    //-------------------------------------------------

    hipMemcpy(rho, d_rho, N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++)
    {
      cout << "AAA = " << rho[i] << endl;
    }

    //------------ SAVING SNAP-SHOTS ------------
    if (!(counter % 20))
    {
      hipMemcpy(Typ, d_Typ, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(vx, d_vx, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vy, d_vy, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vz, d_vz, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(rho, d_rho, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(h, d_h, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(u, d_u, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(mass, d_mass, N * sizeof(float), hipMemcpyDeviceToHost);

      // Specify the output file name
      std::string filename = "./Outputs/G-" + to_string(t * 1) + ".bin";
      // Save the arrays to binary format
      saveArraysToBinary(filename, x, y, z, vx, vy, vz, rho, h, u, mass, Typ, N);
    }

    auto T_NG = std::chrono::high_resolution_clock::now();
    // hipMemcpy(d_NGroupz, NGroupz, NG * sizeof(int), hipMemcpyHostToDevice);
    auto end_NG = std::chrono::high_resolution_clock::now();
    auto elapsed_NG = std::chrono::duration_cast<std::chrono::nanoseconds>(end_NG - T_NG);
    cout << "T_NG = " << elapsed_NG.count() * 1e-9 << endl;

    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

    cout << "Elapsed time = " << elapsed.count() * 1e-9 << endl;
    cout << endl;

    //******************************************************
    //************* Updating Time-step dt ******************
    //******************************************************

    dt_array_indiv_dt<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z,
                                               d_vx, d_vy, d_vz,
                                               d_accx, d_accy, d_accz,
                                               d_accx_tot, d_accy_tot, d_accz_tot,
                                               d_h, d_csnd, d_dt_particles,
                                               d_abs_acc_g, d_abs_acc_tot,
                                               d_divV, d_dh_dt, C_CFL,
                                               visc_alpha, d_eps, N);
    hipDeviceSynchronize();

    hipMemcpy(dt_particles, d_dt_particles, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Typ, d_Typ, N * sizeof(float), hipMemcpyDeviceToHost);

    t += dt;

    dt = min_finder(Typ, dt_particles, N);

    if (!(counter % 1))
    {
      cout << "Adopted dt = " << dt << endl;
      cout << "current t = " << t << endl;
      cout << "*****************************" << endl;
      cout << endl;
    }

    counter++;
  }

  delete[] Typ;
  delete[] x;
  delete[] y;
  delete[] z;
  delete[] vx;
  delete[] vy;
  delete[] vz;
  delete[] mass;
  delete[] h;
  delete[] rho;
  delete[] accx;
  delete[] accy;
  delete[] accz;
  delete[] eps;
  delete[] P;
  delete[] csnd;
  delete[] divV;
  delete[] curlV;
  delete[] accx_sph;
  delete[] accy_sph;
  delete[] accz_sph;
  delete[] accx_tot;
  delete[] accy_tot;
  delete[] accz_tot;
  delete[] abs_acc_g;
  delete[] abs_acc_tot;
  delete[] v_sig;
  delete[] dh_dt;
  delete[] u;
  delete[] dudt;
  delete[] utprevious;

  hipFree(d_Typ);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_vx);
  hipFree(d_vy);
  hipFree(d_vz);
  hipFree(d_mass);
  hipFree(d_h);
  hipFree(d_rho);
  hipFree(d_accx);
  hipFree(d_accy);
  hipFree(d_accz);
  hipFree(d_P);
  hipFree(d_csnd);
  hipFree(d_divV);
  hipFree(d_curlV);
  hipFree(d_accx_sph);
  hipFree(d_accy_sph);
  hipFree(d_accz_sph);
  hipFree(d_accx_tot);
  hipFree(d_accy_tot);
  hipFree(d_accz_tot);
  hipFree(d_abs_acc_g);
  hipFree(d_abs_acc_tot);
  hipFree(d_v_sig);
  hipFree(d_dh_dt);
  hipFree(d_u);
  hipFree(d_dudt);
  hipFree(d_utprevious);
}
