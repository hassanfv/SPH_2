#include "hip/hip_runtime.h"
%%writefile test.cu
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <random>
#include <cmath>

using namespace std;

#define tileSize 3


__global__ void matMultX(float *A, float *B, float *C, int N)
{

  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  
  __shared__ float shareA[tileSize * tileSize];
  __shared__ float shareB[tileSize * tileSize];
  
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  float temp = 0.0;
  
  for (int i = 0; i < N/tileSize; i++)
  {
    shareA[ty * tileSize + tx] = A[row * N + (i * tileSize + threadIdx.x)];
    shareB[ty * tileSize + tx] = B[(i * tileSize + threadIdx.y) * N + col];
    __syncthreads();
    
    for (int k = 0; k < tileSize; k++)
    {
      temp += shareA[ty * tileSize + k] * shareB[k * tileSize + tx];
    }
    __syncthreads();
  }
  
  C[row * N + col] = temp;  
}



__global__ void matMult(float *A, float *B, float *C, int N)
{

  int row = blockIdx.x;
  int col = blockIdx.y;
  
  int k = threadIdx.x;
  
  extern __shared__ float smem[];
  
  float temp = A[row * N + k] * B[k * N + col];
  
  smem[k] = temp;
  __syncthreads();
  
  float s = 0.0f;
  
  if (threadIdx.x == 0)
  {
    for (int i = 0; i < N; i++)
    {
      s += smem[i];
    }
    __syncthreads();
    
    C[row * N + col] = s;
  }
  
}


mt19937 gen(42);
uniform_int_distribution<> dist(1, 9);

float generateRandomFloat()
{
  return static_cast<float>(dist(gen));
}



void create_matrix(float *A, int N)
{
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      A[i * N + j] = generateRandomFloat();
    }
  }
}



void print_matrix(float *D, int N)
{
  for (int i = 1; i <= N * N; i++)
  {
    cout << D[i-1] << "   ";
    
    if ((i % N) == 0)
      {
        cout << endl;
      }
  }
}



int main()
{

  int N = 12;

  float *A, *B;
  
  A = new float[N * N];
  B = new float[N * N];
  
  float *d_A, *d_B, *d_C;
  
  hipMalloc((void**)&d_A, N * N * sizeof(float));
  hipMalloc((void**)&d_B, N * N * sizeof(float));
  hipMalloc((void**)&d_C, N * N * sizeof(float));
  
  create_matrix(A, N);
  create_matrix(B, N);
  
  hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 blockSize(tileSize, tileSize, 1);
  dim3 gridSize(N / tileSize, N / tileSize, 1);
  
  //matMult<<<gridSize, blockSize, N * sizeof(float)>>>(d_A, d_B, d_C, N);
  matMultX<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
  
  float *C;
  C = new float[N * N];
  
  hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
  
  /*
  print_matrix(A, N);
  cout << endl << endl;
  
  print_matrix(B, N);
  cout << endl << endl;
  */
  
  print_matrix(C, N);
  cout << endl << endl;
  
  /*
  for (int i = 0; i < N*N; i++)
  {
    cout << C[i] << endl;
  }
  */
  



}




