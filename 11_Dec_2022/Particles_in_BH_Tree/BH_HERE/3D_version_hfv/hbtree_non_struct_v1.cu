#include "hip/hip_runtime.h"
%%writefile test.cu

#include <iostream>
#include <cmath>
#include <random>
#include <fstream>
#include <chrono>
#include <cstdlib>

using namespace std;

const int BLOCK_SIZE = 256;

#define COLLISION_TH 0.05
#define E 0.1
#define GRAVITY 1.0
#define THETA 0.8

//------ Vector -------
struct Vector
{
  float x;
  float y;
  float z;
};


//------ Body -------
struct Body
{
  bool isDynamic = true;
  float mass;
  float radius;
  Vector position;
  Vector velocity;
  Vector acceleration;
};


//------ Node -------
struct Node
{
  Vector minCorner; // Minimum corner of the bounding box
  Vector maxCorner; // Maximum corner of the bounding box
  Vector centerMass;
  float totalMass;
  bool isLeaf;
  int start;
  int end;
};


/*
----------------------------------------------------------------------------------------
RESET KERNEL
----------------------------------------------------------------------------------------
*/
__global__ void ResetKernel(Node *node, int *mutex, int nNodes, int nBodies)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;

    if (b < nNodes)
    {
        node[b].minCorner = {INFINITY, INFINITY, INFINITY};
        node[b].maxCorner = {-INFINITY, -INFINITY, -INFINITY};
        node[b].centerMass = {-1, -1, -1};
        node[b].totalMass = 0.0;
        node[b].isLeaf = true;
        node[b].start = -1;
        node[b].end = -1;
        mutex[b] = 0;
    }

    if (b == 0)
    {
        node[b].start = 0;
        node[b].end = nBodies - 1;
    }
}



/*
----------------------------------------------------------------------------------------
COMPUTE BOUNDING BOX
----------------------------------------------------------------------------------------
*/
__global__ void ComputeBoundingBoxKernel(Node *node, float *d_x, float *d_y, float *d_z, int *mutex, int nBodies)
{
    __shared__ float minX[BLOCK_SIZE];
    __shared__ float minY[BLOCK_SIZE];
    __shared__ float minZ[BLOCK_SIZE];
    __shared__ float maxX[BLOCK_SIZE];
    __shared__ float maxY[BLOCK_SIZE];
    __shared__ float maxZ[BLOCK_SIZE];

    int tx = threadIdx.x;
    int b = tx + blockIdx.x * blockDim.x;

    minX[tx] = INFINITY;
    minY[tx] = INFINITY;
    minZ[tx] = INFINITY;
    maxX[tx] = -INFINITY;
    maxY[tx] = -INFINITY;
    maxZ[tx] = -INFINITY;

    __syncthreads();

    if (b < nBodies)
    {        
        float x = d_x[b];
        float y = d_y[b];
        float z = d_z[b];
        
        minX[tx] = x;
        minY[tx] = y;
        minZ[tx] = z;
        maxX[tx] = x;
        maxY[tx] = y;
        maxZ[tx] = z;
    }

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tx < s)
        {
            minX[tx] = fminf(minX[tx], minX[tx + s]);
            minY[tx] = fminf(minY[tx], minY[tx + s]);
            minZ[tx] = fminf(minZ[tx], minZ[tx + s]);
            maxX[tx] = fmaxf(maxX[tx], maxX[tx + s]);
            maxY[tx] = fmaxf(maxY[tx], maxY[tx + s]);
            maxZ[tx] = fmaxf(maxZ[tx], maxZ[tx + s]);
        }
    }

    if (tx == 0)
    {
        while (atomicCAS(mutex, 0, 1) != 0)
            ;
        node[0].minCorner.x = fminf(node[0].minCorner.x, minX[0] - 0.01);
        node[0].minCorner.y = fminf(node[0].minCorner.y, minY[0] - 0.01);
        node[0].minCorner.z = fminf(node[0].minCorner.z, minZ[0] - 0.01);
        node[0].maxCorner.x = fmaxf(node[0].maxCorner.x, maxX[0] + 0.01);
        node[0].maxCorner.y = fmaxf(node[0].maxCorner.y, maxY[0] + 0.01);
        node[0].maxCorner.z = fmaxf(node[0].maxCorner.z, maxZ[0] + 0.01);
        atomicExch(mutex, 0);
    }
}




/*
----------------------------------------------------------------------------------------
CONSTRUCT QUAD TREE
----------------------------------------------------------------------------------------
*/
__device__ int getOctant(Vector minCorner, Vector maxCorner, float x, float y, float z)
{
    // Calculate the center of the bounding box
    float centerX = (minCorner.x + maxCorner.x) / 2;
    float centerY = (minCorner.y + maxCorner.y) / 2;
    float centerZ = (minCorner.z + maxCorner.z) / 2;

    int octant = 0;

    // Determine the octant
    if (x <= centerX) {
        if (y <= centerY) {
            if (z <= centerZ) {
                octant = 1; // Lower-left-back
            } else {
                octant = 2; // Lower-left-front
            }
        } else {
            if (z <= centerZ) {
                octant = 3; // Upper-left-back
            } else {
                octant = 4; // Upper-left-front
            }
        }
    } else {
        if (y <= centerY) {
            if (z <= centerZ) {
                octant = 5; // Lower-right-back
            } else {
                octant = 6; // Lower-right-front
            }
        } else {
            if (z <= centerZ) {
                octant = 7; // Upper-right-back
            } else {
                octant = 8; // Upper-right-front
            }
        }
    }

    return octant;
}



__device__ void UpdateChildBound(Vector &minCorner, Vector &maxCorner, Node &childNode, int octant)
{
    // Calculate the center of the bounding box
    float centerX = (minCorner.x + maxCorner.x) / 2;
    float centerY = (minCorner.y + maxCorner.y) / 2;
    float centerZ = (minCorner.z + maxCorner.z) / 2;

    // Update the bounds based on the octant
    if (octant == 1) { // Lower-left-back
        childNode.minCorner = {minCorner.x, minCorner.y, minCorner.z};
        childNode.maxCorner = {centerX, centerY, centerZ};
    } else if (octant == 2) { // Lower-left-front
        childNode.minCorner = {minCorner.x, minCorner.y, centerZ};
        childNode.maxCorner = {centerX, centerY, maxCorner.z};
    } else if (octant == 3) { // Upper-left-back
        childNode.minCorner = {minCorner.x, centerY, minCorner.z};
        childNode.maxCorner = {centerX, maxCorner.y, centerZ};
    } else if (octant == 4) { // Upper-left-front
        childNode.minCorner = {minCorner.x, centerY, centerZ};
        childNode.maxCorner = {centerX, maxCorner.y, maxCorner.z};
    } else if (octant == 5) { // Lower-right-back
        childNode.minCorner = {centerX, minCorner.y, minCorner.z};
        childNode.maxCorner = {maxCorner.x, centerY, centerZ};
    } else if (octant == 6) { // Lower-right-front
        childNode.minCorner = {centerX, minCorner.y, centerZ};
        childNode.maxCorner = {maxCorner.x, centerY, maxCorner.z};
    } else if (octant == 7) { // Upper-right-back
        childNode.minCorner = {centerX, centerY, minCorner.z};
        childNode.maxCorner = {maxCorner.x, maxCorner.y, centerZ};
    } else if (octant == 8) { // Upper-right-front
        childNode.minCorner = {centerX, centerY, centerZ};
        childNode.maxCorner = {maxCorner.x, maxCorner.y, maxCorner.z};
    }
}


__device__ void warpReduce(volatile float *totalMass, volatile float3 *centerMass, int tx)
{
    totalMass[tx] += totalMass[tx + 32];
    centerMass[tx].x += centerMass[tx + 32].x;
    centerMass[tx].y += centerMass[tx + 32].y;
    centerMass[tx].z += centerMass[tx + 32].z;

    totalMass[tx] += totalMass[tx + 16];
    centerMass[tx].x += centerMass[tx + 16].x;
    centerMass[tx].y += centerMass[tx + 16].y;
    centerMass[tx].z += centerMass[tx + 16].z;

    totalMass[tx] += totalMass[tx + 8];
    centerMass[tx].x += centerMass[tx + 8].x;
    centerMass[tx].y += centerMass[tx + 8].y;
    centerMass[tx].z += centerMass[tx + 8].z;

    totalMass[tx] += totalMass[tx + 4];
    centerMass[tx].x += centerMass[tx + 4].x;
    centerMass[tx].y += centerMass[tx + 4].y;
    centerMass[tx].z += centerMass[tx + 4].z;

    totalMass[tx] += totalMass[tx + 2];
    centerMass[tx].x += centerMass[tx + 2].x;
    centerMass[tx].y += centerMass[tx + 2].y;
    centerMass[tx].z += centerMass[tx + 2].z;

    totalMass[tx] += totalMass[tx + 1];
    centerMass[tx].x += centerMass[tx + 1].x;
    centerMass[tx].y += centerMass[tx + 1].y;
    centerMass[tx].z += centerMass[tx + 1].z;
}



__device__ void ComputeCenterMass(Node &curNode, float *d_x, float *d_y, float *d_z, float *d_x_buffer, float *d_y_buffer, float *d_z_buffer,
                                  float *d_mass, float *totalMass, float3 *centerMass, int start, int end)
{
    int tx = threadIdx.x;
    int total = end - start + 1;
    int sz = ceil((float)total / blockDim.x);
    int s = tx * sz + start;
    float M = 0.0;
    float3 R = make_float3(0.0, 0.0, 0.0);

    for (int i = s; i < s + sz; ++i)
    {
        if (i <= end)
        {
            float x = d_x[i];
            float y = d_y[i];
            float z = d_z[i];
            
            M += d_mass[i];
            
            R.x += d_mass[i] * x;
            R.y += d_mass[i] * y;
            R.z += d_mass[i] * z;
        }
    }

    totalMass[tx] = M;
    centerMass[tx] = R;

    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            totalMass[tx] += totalMass[tx + stride];
            centerMass[tx].x += centerMass[tx + stride].x;
            centerMass[tx].y += centerMass[tx + stride].y;
            centerMass[tx].z += centerMass[tx + stride].z;
        }
    }

    if (tx < 32)
    {
        warpReduce(totalMass, centerMass, tx);
    }
    __syncthreads();

    if (tx == 0)
    {
        centerMass[0].x /= totalMass[0];
        centerMass[0].y /= totalMass[0];
        centerMass[0].z /= totalMass[0];
        curNode.totalMass = totalMass[0];
        curNode.centerMass = {centerMass[0].x, centerMass[0].y, centerMass[0].z};
    }
}




__device__ void CountBodies(float *d_x, float *d_y, float *d_z, Vector minCorner, Vector maxCorner, int *count, int start, int end, int nBodies)
{
    int tx = threadIdx.x;
    if (tx < 8) // There are 8 octants in 3D space
        count[tx] = 0;
    __syncthreads();

    for (int i = start + tx; i <= end; i += blockDim.x)
    {
        float x = d_x[i];
        float y = d_y[i];
        float z = d_z[i];
        
        int octant = getOctant(minCorner, maxCorner, x, y, z);
        atomicAdd(&count[octant - 1], 1);
    }

    __syncthreads();
}




__device__ void ComputeOffset(int *count, int start)
{
    int tx = threadIdx.x;
    if (tx < 8) // For 8 octants in 3D space
    {
        int offset = start;
        for (int i = 0; i < tx; ++i)
        {
            offset += count[i];
        }
        count[tx + 8] = offset; // Store the offset for each octant
    }
    __syncthreads();
}




__device__ void GroupBodies(float *d_x, float *d_y, float *d_z, float *d_x_buffer, float *d_y_buffer, float *d_z_buffer, int *d_refIndex,
                            Vector minCorner, Vector maxCorner, int *count, int start, int end, int nBodies)
{
  int *offsets = &count[8]; // Updated for 8 octants
  for (int i = start + threadIdx.x; i <= end; i += blockDim.x)
  {
    if (i < nBodies)
    {
      float x = d_x[i];
      float y = d_y[i];
      float z = d_z[i];

      int octant = getOctant(minCorner, maxCorner, x, y, z) - 1;
      int dest = atomicAdd(&offsets[octant], 1);
      
      d_x_buffer[dest] = x;
      d_y_buffer[dest] = y;
      d_z_buffer[dest] = z;
      
      d_refIndex[i] = dest;
    }
  }
  __syncthreads();
}




__global__ void ConstructQuadTreeKernel(Node *node, float *d_x, float *d_y, float *d_z, float *d_x_buffer, float *d_y_buffer, float *d_z_buffer,
                                        int *d_refIndex, float *d_mass, int nodeIndex, int nNodes, int nBodies, int leafLimit)
{
    __shared__ int count[16];
    __shared__ float totalMass[BLOCK_SIZE];
    __shared__ float3 centerMass[BLOCK_SIZE];
    int tx = threadIdx.x;
    nodeIndex += blockIdx.x;

    if (nodeIndex > 990000)
      printf("nodeIndex, blockIdx.x = %d, %d\n", nodeIndex, blockIdx.x);

    if (nodeIndex >= nNodes)
        return;

    Node &curNode = node[nodeIndex];
    int start = curNode.start;
    int end = curNode.end;
    
    Vector minCorner = curNode.minCorner;
    Vector maxCorner = curNode.maxCorner;

    if (start == -1 && end == -1)
        return;
    
    if (end - start < 50) //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
      return;

    ComputeCenterMass(curNode, d_x, d_y, d_z, d_x_buffer, d_y_buffer, d_z_buffer, d_mass, totalMass, centerMass, start, end);
    if (nodeIndex >= leafLimit || start == end)
    {
        for (int i = start; i <= end; ++i)
        {            
            d_x_buffer[i] = d_x[i];
            d_y_buffer[i] = d_y[i];
            d_z_buffer[i] = d_z[i];
        }

        return;
    }

    CountBodies(d_x, d_y, d_z, minCorner, maxCorner, count, start, end, nBodies);
    ComputeOffset(count, start);
    GroupBodies(d_x, d_y, d_z, d_x_buffer, d_y_buffer, d_z_buffer, d_refIndex, minCorner, maxCorner, count, start, end, nBodies);

    if (tx == 0)
    {
        Node &LLBNode = node[(nodeIndex * 8) + 1];
        Node &LLFNode = node[(nodeIndex * 8) + 2];
        Node &ULBNode = node[(nodeIndex * 8) + 3];
        Node &ULFNode = node[(nodeIndex * 8) + 4];
        Node &LRBNode = node[(nodeIndex * 8) + 5];
        Node &LRFNode = node[(nodeIndex * 8) + 6];
        Node &URBNode = node[(nodeIndex * 8) + 7];
        Node &URFNode = node[(nodeIndex * 8) + 8];
    
        UpdateChildBound(minCorner, maxCorner, LLBNode, 1);
        UpdateChildBound(minCorner, maxCorner, LLFNode, 2);
        UpdateChildBound(minCorner, maxCorner, ULBNode, 3);
        UpdateChildBound(minCorner, maxCorner, ULFNode, 4);
        UpdateChildBound(minCorner, maxCorner, LRBNode, 5);
        UpdateChildBound(minCorner, maxCorner, LRFNode, 6);
        UpdateChildBound(minCorner, maxCorner, URBNode, 7);
        UpdateChildBound(minCorner, maxCorner, URFNode, 8);

        curNode.isLeaf = false;
        
        if (count[0] > 0)
        {
            LLBNode.start = start;
            LLBNode.end = start + count[0] - 1;
        }
        
        if (count[1] > 0)
        {
            LLFNode.start = start + count[0];
            LLFNode.end = start + count[0] + count[1] - 1;
        }
        
        if (count[2] > 0)
        {
            ULBNode.start = start + count[0] + count[1];
            ULBNode.end = start + count[0] + count[1] + count[2] - 1;
        }
        
        if (count[3] > 0)
        {
            ULFNode.start = start + count[0] + count[1] + count[2];
            ULFNode.end = start + count[0] + count[1] + count[2] + count[3] - 1;
        }
        
        if (count[4] > 0)
        {
            LRBNode.start = start + count[0] + count[1] + count[2] + count[3];
            LRBNode.end = start + count[0] + count[1] + count[2] + count[3] + count[4] - 1;
        }
        
        if (count[5] > 0)
        {
            LRFNode.start = start + count[0] + count[1] + count[2] + count[3] + count[4];
            LRFNode.end = start + count[0] + count[1] + count[2] + count[3] + count[4] + count[5] - 1;
        }
        
        if (count[6] > 0)
        {
            URBNode.start = start + count[0] + count[1] + count[2] + count[3] + count[4] + count[5];
            URBNode.end = start + count[0] + count[1] + count[2] + count[3] + count[4] + count[5] + count[6] - 1;
        }
        
        if (count[7] > 0)
        {
            URFNode.start = start + count[0] + count[1] + count[2] + count[3] + count[4] + count[5] + count[6];
            URFNode.end = end;
        }
        ConstructQuadTreeKernel<<<8, BLOCK_SIZE>>>(node, d_x_buffer, d_y_buffer, d_z_buffer, d_x, d_y, d_z, d_refIndex,
                                                   d_mass, nodeIndex * 8 + 1, nNodes, nBodies, leafLimit);
    }
}



/*
----------------------------------------------------------------------------------------
COMPUTE FORCE
----------------------------------------------------------------------------------------
*/
__device__ float getDistance(float x, float y, float z, Vector cm)
{

    return sqrt(pow(x - cm.x, 2) + pow(y - cm.y, 2) + pow(z - cm.z, 2));
}

__device__ bool isCollide(float x, float y, float z, Vector cm)
{
    return 0.1 * 2 + COLLISION_TH > getDistance(x, y, z, cm);
}


// Note that the d_x, d_y, d_z that goes into this function h gone through the buffer stage so ordering is disruppted. So we need refIndex for them!!!
__device__ void ComputeForce(Node *node, float *d_x, float *d_y, float *d_z, float *d_acc_g_x, float *d_acc_g_y, float *d_acc_g_z, int *d_refIndex,
                             int nodeIndex, int bodyIndex, int nNodes, int nBodies, int leafLimit, float width)
{

    if (nodeIndex >= nNodes)
    {
        return;
    }
    Node curNode = node[nodeIndex];
    
    bodyIndex = d_refIndex[bodyIndex]; // due to the exchange occured in buffer arrays we need to do this for d_x, d_y, d_z.
    
    float x = d_x[bodyIndex];
    float y = d_y[bodyIndex];
    float z = d_z[bodyIndex];
    
    if (curNode.isLeaf)
    {
        if (curNode.centerMass.x != -1 && !isCollide(x, y, z, curNode.centerMass))
        {
            Vector rij = {curNode.centerMass.x - x, curNode.centerMass.y - y, curNode.centerMass.z - z};
            float r = sqrt((rij.x * rij.x) + (rij.y * rij.y) + (rij.z * rij.z) + (E * E));
            float f = (GRAVITY * curNode.totalMass) / (r * r * r + (E * E));
            Vector force = {rij.x * f, rij.y * f, rij.z * f};
            
            d_acc_g_x[bodyIndex] += (force.x);
            d_acc_g_y[bodyIndex] += (force.y);
            d_acc_g_z[bodyIndex] += (force.z);
        }
        return;
    }

    float sd = width / getDistance(x, y, z, curNode.centerMass);
    if (sd < THETA)
    {
        if (!isCollide(x, y, z, curNode.centerMass))
        {
            Vector rij = {curNode.centerMass.x - x, curNode.centerMass.y - y, curNode.centerMass.z - z};
            float r = sqrt((rij.x * rij.x) + (rij.y * rij.y) + (rij.z * rij.z) + (E * E));
            float f = (GRAVITY * curNode.totalMass) / (r * r * r + (E * E));
            Vector force = {rij.x * f, rij.y * f, rij.z * f};

            d_acc_g_x[bodyIndex] += (force.x);
            d_acc_g_y[bodyIndex] += (force.y);
            d_acc_g_z[bodyIndex] += (force.z);
        }

        return;
    }

    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 1, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 2, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 3, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 4, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 5, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 6, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 7, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, (nodeIndex * 8) + 8, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
}



__global__ void ComputeForceKernel(Node *node, float *d_x, float *d_y, float *d_z, float *d_acc_g_x, float *d_acc_g_y, float *d_acc_g_z, int *d_refIndex,
                                   int nNodes, int nBodies, int leafLimit)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float width = node[0].maxCorner.x - node[0].minCorner.x;
    
    if (i < nBodies)
    {   
        if (true)
        {
            d_acc_g_x[i] = 0.0;
            d_acc_g_y[i] = 0.0;
            d_acc_g_z[i] = 0.0;
            
            ComputeForce(node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, 0, i, nNodes, nBodies, leafLimit, width);
        }
    }
}





void saveToFile(const char* filename, Body* h_b, Node* h_node, int nBodies, int nNodes) {
    ofstream file(filename, ios::out | ios::binary);
    if (!file.is_open()) {
        cerr << "Error opening file for writing!" << endl;
        return;
    }

    // Write nBodies and nNodes
    file.write(reinterpret_cast<char*>(&nBodies), sizeof(nBodies));
    file.write(reinterpret_cast<char*>(&nNodes), sizeof(nNodes));

    // Write positions of bodies
    for (int i = 0; i < nBodies; ++i) {
        file.write(reinterpret_cast<char*>(&h_b[i].position.x), sizeof(h_b[i].position.x));
        file.write(reinterpret_cast<char*>(&h_b[i].position.y), sizeof(h_b[i].position.y));
        file.write(reinterpret_cast<char*>(&h_b[i].position.z), sizeof(h_b[i].position.z));
    }

    // Write boundaries of nodes
    for (int i = 0; i < nNodes; ++i) {
        file.write(reinterpret_cast<char*>(&h_node[i].minCorner.x), sizeof(h_node[i].minCorner.x));
        file.write(reinterpret_cast<char*>(&h_node[i].minCorner.y), sizeof(h_node[i].minCorner.y));
        file.write(reinterpret_cast<char*>(&h_node[i].minCorner.z), sizeof(h_node[i].minCorner.z));
        file.write(reinterpret_cast<char*>(&h_node[i].maxCorner.x), sizeof(h_node[i].maxCorner.x));
        file.write(reinterpret_cast<char*>(&h_node[i].maxCorner.y), sizeof(h_node[i].maxCorner.y));
        file.write(reinterpret_cast<char*>(&h_node[i].maxCorner.z), sizeof(h_node[i].maxCorner.z));
        file.write(reinterpret_cast<char*>(&h_node[i].start), sizeof(h_node[i].start));
        file.write(reinterpret_cast<char*>(&h_node[i].end), sizeof(h_node[i].end));
    }

    file.close();
}




void saveBodyToFile(const char* filename, Body* h_b, int nBodies)
{
    ofstream file(filename, ios::out | ios::binary);
    if (!file.is_open()) {
        cerr << "Error opening file for writing!" << endl;
        return;
    }

    // Write nBodies
    file.write(reinterpret_cast<char*>(&nBodies), sizeof(nBodies));

    // Write bodies
    for (int i = 0; i < nBodies; ++i)
    {
        file.write(reinterpret_cast<char*>(&h_b[i]), sizeof(Body));
    }

    file.close();
}




int main()
{

  int n = 1000000; // number of particles.
  int nBodies = n;

  int MAX_NODES = 1000000;
  int N_LEAF = 500000;

  int nNodes = MAX_NODES;
  int leafLimit = MAX_NODES - N_LEAF;
  
  float *x, *y, *z, *d_x, *d_y, *d_z, *acc_g_x, *acc_g_y, *acc_g_z, *d_acc_g_x, *d_acc_g_y, *d_acc_g_z;
  float *mass, *d_mass, *x_buffer, *y_buffer, *z_buffer, *d_x_buffer, *d_y_buffer, *d_z_buffer;
  int *refIndex, *d_refIndex;
  
  //Body *h_b, *d_b, *d_b_buffer;
  Node *h_node, *d_node;
  
  int *d_mutex;
  
  //h_b = new Body[n];
  
  x = new float[nBodies];
  y = new float[nBodies];
  z = new float[nBodies];
  
  x_buffer = new float[nBodies];
  y_buffer = new float[nBodies];
  z_buffer = new float[nBodies];
  
  acc_g_x = new float[nBodies];
  acc_g_y = new float[nBodies];
  acc_g_z = new float[nBodies];
  
  mass = new float[nBodies];
  
  refIndex = new int[nBodies];
  
  h_node = new Node[nNodes];

  hipMalloc((void **)&d_x, sizeof(float) * nBodies);
  hipMalloc((void **)&d_y, sizeof(float) * nBodies);
  hipMalloc((void **)&d_z, sizeof(float) * nBodies);
  
  hipMalloc((void **)&d_mass, sizeof(float) * nBodies);
  
  hipMalloc((void **)&d_refIndex, sizeof(int) * nBodies);
  
  hipMalloc((void **)&d_x_buffer, sizeof(float) * nBodies);
  hipMalloc((void **)&d_y_buffer, sizeof(float) * nBodies);
  hipMalloc((void **)&d_z_buffer, sizeof(float) * nBodies);
  
  hipMalloc((void **)&d_acc_g_x, sizeof(float) * nBodies);
  hipMalloc((void **)&d_acc_g_y, sizeof(float) * nBodies);
  hipMalloc((void **)&d_acc_g_z, sizeof(float) * nBodies);


  hipMalloc((void **)&d_node, sizeof(Node) * nNodes);
  hipMalloc((void **)&d_mutex, sizeof(int) * nNodes);

  //--- preparing bodies (bodies are actually particles!) ---
  mt19937 eng(42); // Seed the generator
  uniform_real_distribution<> distr(-1.0, 1.0); // Define the range
  
  for (int i = 0; i < n; i++)
  {
    x[i] = distr(eng);
    y[i] = distr(eng);
    z[i] = distr(eng);
    
    x_buffer[i] = 0.0;
    y_buffer[i] = 0.0;
    z_buffer[i] = 0.0;
    
    acc_g_x[i] = 0.0;
    acc_g_y[i] = 0.0;
    acc_g_z[i] = 0.0;
    
    mass[i] = 1.0;
    
    refIndex[i] = i;
  
  }
  
  
  int i = 100;
  cout << "x = " << x[i] << endl;
  cout << "y = " << y[i] << endl;
  cout << "z = " << z[i] << endl;
  cout << endl;
  
  
  //--- copying particles from host to device
  hipMemcpy(d_x, x, nBodies * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, nBodies * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, nBodies * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_x_buffer, x_buffer, nBodies * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y_buffer, y_buffer, nBodies * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z_buffer, z_buffer, nBodies * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_mass, mass, nBodies * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_acc_g_x, acc_g_x, nBodies * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_acc_g_y, acc_g_y, nBodies * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_acc_g_z, acc_g_z, nBodies * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_refIndex, refIndex, nBodies * sizeof(int), hipMemcpyHostToDevice);
  
  cout << "h_node.minCorner.x = " << h_node[0].minCorner.x << endl;
  cout << "h_node.minCorner.y = " << h_node[0].minCorner.y << endl;
  cout << "h_node.minCorner.z = " << h_node[0].minCorner.z << endl;
  cout << "h_node.maxCorner.x = " << h_node[0].maxCorner.x << endl;
  cout << "h_node.maxCorner.y = " << h_node[0].maxCorner.y << endl;
  cout << "h_node.maxCorner.z = " << h_node[0].maxCorner.z << endl;
  
  cout << "h_node.start = " << h_node[0].start << endl;
  cout << "h_node.end = " << h_node[0].end << endl;
  cout << endl;
  
  int blockSize = BLOCK_SIZE;
  dim3 gridSize = ceil((float)nNodes / blockSize);
  ResetKernel<<<gridSize, blockSize>>>(d_node, d_mutex, nNodes, nBodies);
  hipDeviceSynchronize();

  hipMemcpy(h_node, d_node, nNodes * sizeof(Node), hipMemcpyDeviceToHost);
  cout << "h_node.minCorner.x = " << h_node[0].minCorner.x << endl;
  cout << "h_node.minCorner.y = " << h_node[0].minCorner.y << endl;
  cout << "h_node.minCorner.z = " << h_node[0].minCorner.z << endl;
  cout << "h_node.maxCorner.x = " << h_node[0].maxCorner.x << endl;
  cout << "h_node.maxCorner.y = " << h_node[0].maxCorner.y << endl;
  cout << "h_node.maxCorner.z = " << h_node[0].maxCorner.z << endl;
  
  cout << "h_node.start = " << h_node[0].start << endl;
  cout << "h_node.end = " << h_node[0].end << endl;
  cout << endl;


  blockSize = BLOCK_SIZE;
  gridSize = ceil((float)nBodies / blockSize);
  ComputeBoundingBoxKernel<<<gridSize, blockSize>>>(d_node, d_x, d_y, d_z, d_mutex, nBodies);
  hipDeviceSynchronize();

  hipMemcpy(h_node, d_node, nNodes * sizeof(Node), hipMemcpyDeviceToHost);
  cout << "h_node.minCorner.x = " << h_node[0].minCorner.x << endl;
  cout << "h_node.minCorner.y = " << h_node[0].minCorner.y << endl;
  cout << "h_node.minCorner.z = " << h_node[0].minCorner.z << endl;
  cout << "h_node.maxCorner.x = " << h_node[0].maxCorner.x << endl;
  cout << "h_node.maxCorner.y = " << h_node[0].maxCorner.y << endl;
  cout << "h_node.maxCorner.z = " << h_node[0].maxCorner.z << endl;
  
  cout << "h_node.start = " << h_node[0].start << endl;
  cout << "h_node.end = " << h_node[0].end << endl;
  cout << endl;
  
  blockSize = BLOCK_SIZE;
  gridSize = ceil((float)nBodies / blockSize);
  ConstructQuadTreeKernel<<<1, blockSize>>>(d_node, d_x, d_y, d_z, d_x_buffer, d_y_buffer, d_z_buffer, d_refIndex, d_mass, 0, nNodes, nBodies, leafLimit);
  hipDeviceSynchronize();
  
  hipMemcpy(x, d_x, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  hipMemcpy(z, d_z, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  
  cout << "x = " << x[i] << endl;
  cout << "y = " << y[i] << endl;
  cout << "z = " << z[i] << endl;
  cout << endl;
  
  exit(0);
  

  //----- example prints --
  hipMemcpy(h_node, d_node, sizeof(Node) * nNodes, hipMemcpyDeviceToHost);
  Node node_1 = h_node[100];
  cout << "start = " << node_1.start << endl;
  cout << "end = " << node_1.end << endl;


  auto T_1 = std::chrono::high_resolution_clock::now();

  blockSize = 32; // perhaps because of the warp use in ComputeForce function!
  gridSize = ceil((float)nBodies / blockSize);
  ComputeForceKernel<<<gridSize, blockSize>>>(d_node, d_x, d_y, d_z, d_acc_g_x, d_acc_g_y, d_acc_g_z, d_refIndex, nNodes, nBodies, leafLimit);
  hipDeviceSynchronize();
  
  auto end_1 = std::chrono::high_resolution_clock::now();
  auto elapsed_1 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_1 - T_1);
  cout << "T_1 = " << elapsed_1.count() * 1e-9 << endl;
  
  hipMemcpy(acc_g_x, d_acc_g_x, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  hipMemcpy(acc_g_y, d_acc_g_y, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  hipMemcpy(acc_g_z, d_acc_g_z, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  
  hipMemcpy(x, d_x, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  hipMemcpy(z, d_z, sizeof(float) * nBodies, hipMemcpyDeviceToHost);
  
  hipMemcpy(refIndex, d_refIndex, sizeof(int) * nBodies, hipMemcpyDeviceToHost);
  

  //int i = 100;

  printf("(accx, accy, accz) = %f, %f, %f\n", acc_g_x[i], acc_g_y[i], acc_g_z[i]);
  cout << "x = " << x[refIndex[i]] << endl;
  cout << "y = " << y[refIndex[i]] << endl;
  cout << "z = " << z[refIndex[i]] << endl;
  
  cout << endl;

  cout << "refIndex[i] = " << refIndex[i] << endl;
  cout << endl;

  /*
  // Save h_b to a binary file
  for (int i = 0; i < n; i++)
  {
    h_b[i].acceleration.x = 0.0;
    h_b[i].acceleration.y = 0.0;
    h_b[i].acceleration.z = 0.0;
  }

  saveBodyToFile("h_b.bin", h_b, nBodies);
  */


  //----- Output to a file -----
  //saveToFile("BH.bin", h_b, h_node, nBodies, nNodes);
  


}



